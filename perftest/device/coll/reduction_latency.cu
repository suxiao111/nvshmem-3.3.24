#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "reduction_latency.cubin"

#include "utils.h"
#include "coll_test.h"
#define LARGEST_DT int64_t

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

#define CALL_RDXN(TG_PRE, TG, TYPENAME, TYPE, OP, THREAD_COMP, ELEM_COMP)                         \
                                                                                                  \
    void call_test_##TYPENAME##_##OP##_reduce_kern##TG##_cubin(                                   \
        int num_blocks, int num_tpb, hipStream_t stream, void **arglist) {                       \
        hipFunction_t test_##TYPENAME##_##OP##_reduce_kern##TG_cubin;                                \
                                                                                                  \
        init_test_case_kernel(&test_##TYPENAME##_##OP##_reduce_kern##TG_cubin,                    \
                              NVSHMEMI_TEST_STRINGIFY(test_##TYPENAME##_##OP##_reduce_kern##TG)); \
        CU_CHECK(hipModuleLaunchCooperativeKernel(test_##TYPENAME##_##OP##_reduce_kern##TG_cubin,        \
                                           num_blocks, 1, 1, num_tpb, 1, 1, 0, stream, arglist)); \
    }                                                                                             \
                                                                                                  \
    __global__ void test_##TYPENAME##_##OP##_reduce_kern##TG(                                     \
        nvshmem_team_t team, TYPE *dest, const TYPE *source, int nelems, int iter) {              \
        int i;                                                                                    \
                                                                                                  \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP) && (nelems < ELEM_COMP)) {                 \
            for (i = 0; i < iter; i++) {                                                          \
                nvshmem##TG_PRE##_##TYPENAME##_##OP##_reduce##TG(team, dest, source, nelems);     \
            }                                                                                     \
        }                                                                                         \
    }

#define CALL_RDXN_KERNEL(TYPENAME, OP, TG, BLOCKS, THREADS, ARG_LIST, STREAM)                     \
    if (use_cubin) {                                                                              \
        call_test_##TYPENAME##_##OP##_reduce_kern##TG##_cubin(BLOCKS, THREADS, STREAM, ARG_LIST); \
    } else {                                                                                      \
        status =                                                                                  \
            nvshmemx_collective_launch((const void *)test_##TYPENAME##_##OP##_reduce_kern##TG,    \
                                       BLOCKS, THREADS, ARG_LIST, 0, STREAM);                     \
        if (status != NVSHMEMX_SUCCESS) {                                                         \
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);                     \
            exit(-1);                                                                             \
        }                                                                                         \
    }

#define CALL_RDXN_OPS_ALL_TG(TYPENAME, TYPE)                     \
    CALL_RDXN(x, _block, TYPENAME, TYPE, sum, INT_MAX, INT_MAX)  \
    CALL_RDXN(x, _block, TYPENAME, TYPE, prod, INT_MAX, INT_MAX) \
    CALL_RDXN(x, _block, TYPENAME, TYPE, and, INT_MAX, INT_MAX)  \
    CALL_RDXN(x, _block, TYPENAME, TYPE, or, INT_MAX, INT_MAX)   \
    CALL_RDXN(x, _block, TYPENAME, TYPE, xor, INT_MAX, INT_MAX)  \
    CALL_RDXN(x, _block, TYPENAME, TYPE, min, INT_MAX, INT_MAX)  \
    CALL_RDXN(x, _block, TYPENAME, TYPE, max, INT_MAX, INT_MAX)  \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, sum, warpSize, 4096)     \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, prod, warpSize, 4096)    \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, and, warpSize, 4096)     \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, or, warpSize, 4096)      \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, xor, warpSize, 4096)     \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, min, warpSize, 4096)     \
    CALL_RDXN(x, _warp, TYPENAME, TYPE, max, warpSize, 4096)     \
    CALL_RDXN(, , TYPENAME, TYPE, sum, 1, 512)                   \
    CALL_RDXN(, , TYPENAME, TYPE, prod, 1, 512)                  \
    CALL_RDXN(, , TYPENAME, TYPE, and, 1, 512)                   \
    CALL_RDXN(, , TYPENAME, TYPE, or, 1, 512)                    \
    CALL_RDXN(, , TYPENAME, TYPE, xor, 1, 512)                   \
    CALL_RDXN(, , TYPENAME, TYPE, min, 1, 512)                   \
    CALL_RDXN(, , TYPENAME, TYPE, max, 1, 512)

CALL_RDXN_OPS_ALL_TG(int32, int32_t)
CALL_RDXN_OPS_ALL_TG(int64, int64_t)

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

#define SET_SIZE_ARR(TYPE, ELEM_COMP)                                                      \
    do {                                                                                   \
        j = 0;                                                                             \
        for (num_elems = min_elems; num_elems <= max_elems; num_elems *= step_factor) {    \
            if (num_elems < ELEM_COMP) {                                                   \
                size_arr[j] =                                                              \
                    calculate_collective_size("reduction", num_elems, sizeof(TYPE), npes); \
            } else {                                                                       \
                size_arr[j] = 0;                                                           \
            }                                                                              \
            j++;                                                                           \
        }                                                                                  \
    } while (0)

#define RUN_ITERS_OP(TYPENAME, TYPE, GROUP, OP, ELEM_COMP)                                       \
    do {                                                                                         \
        void *skip_arg_list[] = {&team, &dest, &source, &num_elems, &skip};                      \
        void *time_arg_list[] = {&team, &dest, &source, &num_elems, &iter};                      \
        float milliseconds;                                                                      \
        hipEvent_t start, stop;                                                                 \
        hipEventCreate(&start);                                                                 \
        hipEventCreate(&stop);                                                                  \
        SET_SIZE_ARR(TYPE, ELEM_COMP);                                                           \
                                                                                                 \
        nvshmem_barrier_all();                                                                   \
        j = 0;                                                                                   \
        for (num_elems = min_elems; num_elems < ELEM_COMP; num_elems *= 2) {                     \
            CALL_RDXN_KERNEL(TYPENAME, OP, GROUP, num_blocks, nvshm_test_num_tpb, skip_arg_list, \
                             stream);                                                            \
            CUDA_CHECK(hipStreamSynchronize(stream));                                           \
            nvshmem_barrier_all();                                                               \
                                                                                                 \
            hipEventRecord(start, stream);                                                      \
            CALL_RDXN_KERNEL(TYPENAME, OP, GROUP, num_blocks, nvshm_test_num_tpb, time_arg_list, \
                             stream);                                                            \
            hipEventRecord(stop, stream);                                                       \
            CUDA_CHECK(hipStreamSynchronize(stream));                                           \
                                                                                                 \
            if (!mype) {                                                                         \
                hipEventElapsedTime(&milliseconds, start, stop);                                \
                h_##OP##_lat[j] = (milliseconds * 1000.0) / (float)iter;                         \
            }                                                                                    \
            nvshmem_barrier_all();                                                               \
            j++;                                                                                 \
        }                                                                                        \
    } while (0)

#define RUN_ITERS(TYPENAME, TYPE, GROUP, ELEM_COMP)       \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, sum, ELEM_COMP);  \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, prod, ELEM_COMP); \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, and, ELEM_COMP);  \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, or, ELEM_COMP);   \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, xor, ELEM_COMP);  \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, min, ELEM_COMP);  \
    RUN_ITERS_OP(TYPENAME, TYPE, GROUP, max, ELEM_COMP);

int rdxn_calling_kernel(nvshmem_team_t team, void *dest, const void *source, int mype,
                        hipStream_t stream, run_opt_t run_options, void **h_tables) {
    int status = 0;
    int nvshm_test_num_tpb = threads_per_block;
    int num_blocks = 1;
    size_t num_elems = 1, min_elems, max_elems;
    int iter = iters;
    int skip = warmup_iters;
    int j;
    int npes = nvshmem_n_pes();
    uint64_t *size_arr = (uint64_t *)h_tables[0];
    double *h_sum_lat = (double *)h_tables[1];
    double *h_prod_lat = (double *)h_tables[2];
    double *h_and_lat = (double *)h_tables[3];
    double *h_or_lat = (double *)h_tables[4];
    double *h_xor_lat = (double *)h_tables[5];
    double *h_min_lat = (double *)h_tables[6];
    double *h_max_lat = (double *)h_tables[7];

    // if (!mype) printf("Transfer size in bytes and latency of thread/warp/block variants of all
    // operations of reduction API in us\n");
    if (run_options.run_thread) {
        min_elems = max(static_cast<size_t>(1), min_size / sizeof(int32_t));
        max_elems = max(static_cast<size_t>(1), max_size / sizeof(int32_t));
        RUN_ITERS(int32, int32_t, , 512);
        if (!mype) {
            print_table_v1("device_reduction", "int32-sum-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_sum_lat, j);
            print_table_v1("device_reduction", "int32-prod-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_prod_lat, j);
            print_table_v1("device_reduction", "int32-and-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_and_lat, j);
            print_table_v1("device_reduction", "int32-or-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_or_lat, j);
            print_table_v1("device_reduction", "int32-xor-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_xor_lat, j);
            print_table_v1("device_reduction", "int32-min-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_min_lat, j);
            print_table_v1("device_reduction", "int32-max-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_max_lat, j);
        }

        min_elems = max(static_cast<size_t>(1), min_size / sizeof(int64_t));
        max_elems = max(static_cast<size_t>(1), max_size / sizeof(int64_t));
        RUN_ITERS(int64, int64_t, , 512);
        if (!mype) {
            print_table_v1("device_reduction", "int64-sum-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_sum_lat, j);
            print_table_v1("device_reduction", "int64-prod-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_prod_lat, j);
            print_table_v1("device_reduction", "int64-and-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_and_lat, j);
            print_table_v1("device_reduction", "int64-or-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_or_lat, j);
            print_table_v1("device_reduction", "int64-xor-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_xor_lat, j);
            print_table_v1("device_reduction", "int64-min-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_min_lat, j);
            print_table_v1("device_reduction", "int64-max-t", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_max_lat, j);
        }
    }

    if (run_options.run_warp) {
        min_elems = max(static_cast<size_t>(1), min_size / sizeof(int32_t));
        max_elems = max(static_cast<size_t>(1), max_size / sizeof(int32_t));
        RUN_ITERS(int32, int32_t, _warp, 4096);
        if (!mype) {
            print_table_v1("device_reduction", "int32-sum-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_sum_lat, j);
            print_table_v1("device_reduction", "int32-prod-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_prod_lat, j);
            print_table_v1("device_reduction", "int32-and-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_and_lat, j);
            print_table_v1("device_reduction", "int32-or-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_or_lat, j);
            print_table_v1("device_reduction", "int32-xor-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_xor_lat, j);
            print_table_v1("device_reduction", "int32-min-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_min_lat, j);
            print_table_v1("device_reduction", "int32-max-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_max_lat, j);
        }

        min_elems = max(static_cast<size_t>(1), min_size / sizeof(int64_t));
        max_elems = max(static_cast<size_t>(1), max_size / sizeof(int64_t));
        RUN_ITERS(int64, int64_t, _warp, 4096);
        if (!mype) {
            print_table_v1("device_reduction", "int64-sum-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_sum_lat, j);
            print_table_v1("device_reduction", "int64-prod-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_prod_lat, j);
            print_table_v1("device_reduction", "int64-and-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_and_lat, j);
            print_table_v1("device_reduction", "int64-or-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_or_lat, j);
            print_table_v1("device_reduction", "int64-xor-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_xor_lat, j);
            print_table_v1("device_reduction", "int64-min-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_min_lat, j);
            print_table_v1("device_reduction", "int64-max-w", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_max_lat, j);
        }
    }

    if (run_options.run_block) {
        min_elems = max(static_cast<size_t>(1), min_size / sizeof(int32_t));
        max_elems = max(static_cast<size_t>(1), max_size / sizeof(int32_t));
        RUN_ITERS(int32, int32_t, _block, max_elems);
        if (!mype) {
            print_table_v1("device_reduction", "int32-sum-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_sum_lat, j);
            print_table_v1("device_reduction", "int32-prod-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_prod_lat, j);
            print_table_v1("device_reduction", "int32-and-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_and_lat, j);
            print_table_v1("device_reduction", "int32-or-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_or_lat, j);
            print_table_v1("device_reduction", "int32-xor-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_xor_lat, j);
            print_table_v1("device_reduction", "int32-min-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_min_lat, j);
            print_table_v1("device_reduction", "int32-max-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_max_lat, j);
        }

        min_elems = max(static_cast<size_t>(1), min_size / sizeof(int64_t));
        max_elems = max(static_cast<size_t>(1), max_size / sizeof(int64_t));
        RUN_ITERS(int64, int64_t, _block, max_elems);
        if (!mype) {
            print_table_v1("device_reduction", "int64-sum-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_sum_lat, j);
            print_table_v1("device_reduction", "int64-prod-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_prod_lat, j);
            print_table_v1("device_reduction", "int64-and-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_and_lat, j);
            print_table_v1("device_reduction", "int64-or-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_or_lat, j);
            print_table_v1("device_reduction", "int64-xor-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_xor_lat, j);
            print_table_v1("device_reduction", "int64-min-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_min_lat, j);
            print_table_v1("device_reduction", "int64-max-b", "size (Bytes)", "latency", "us", '-',
                           size_arr, h_max_lat, j);
        }
    }

    return status;
}

int main(int argc, char **argv) {
    int status = 0;
    int mype, array_size;
    size_t size = 0;

    read_args(argc, argv);
    int *h_buffer = NULL;
    int *d_source, *d_dest;
    int *h_source, *h_dest;
    char size_string[100];
    hipStream_t cstrm;
    run_opt_t run_options;
    void **h_tables;

    run_options.run_thread = run_options.run_warp = run_options.run_block = 1;

    size = page_size_roundoff(max_size);   // send buf
    size += page_size_roundoff(max_size);  // recv buf

    DEBUG_PRINT("symmetric size requested %lu\n", size);
    sprintf(size_string, "%lu", size);

    status = setenv("NVSHMEM_SYMMETRIC_SIZE", size_string, 1);
    if (status) {
        fprintf(stderr, "setenv failed \n");
        status = -1;
        goto out;
    }

    array_size = max_size_log;

    init_wrapper(&argc, &argv);
    alloc_tables(&h_tables, 8, array_size);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
    }

    mype = nvshmem_my_pe();

    CUDA_CHECK(hipStreamCreateWithFlags(&cstrm, hipStreamNonBlocking));

    CUDA_CHECK(hipHostAlloc(&h_buffer, max_size * 2, hipHostMallocDefault));
    h_source = (int32_t *)h_buffer;
    h_dest = (int32_t *)&h_source[max_size / sizeof(int32_t)];

    d_source = (int32_t *)nvshmem_align(getpagesize(), max_size);
    d_dest = (int32_t *)nvshmem_align(getpagesize(), max_size);

    CUDA_CHECK(hipMemcpyAsync(d_source, h_source, max_size, hipMemcpyHostToDevice, cstrm));
    CUDA_CHECK(hipMemcpyAsync(d_dest, h_dest, max_size, hipMemcpyHostToDevice, cstrm));

    rdxn_calling_kernel(NVSHMEM_TEAM_WORLD, d_dest, d_source, mype, cstrm, run_options, h_tables);

    DEBUG_PRINT("last error = %s\n", hipGetErrorString(hipGetLastError()));

    CUDA_CHECK(hipMemcpyAsync(h_source, d_source, max_size, hipMemcpyDeviceToHost, cstrm));
    CUDA_CHECK(hipMemcpyAsync(h_dest, d_dest, max_size, hipMemcpyDeviceToHost, cstrm));

    nvshmem_barrier_all();

    CUDA_CHECK(hipHostFree(h_buffer));
    nvshmem_free(d_source);
    nvshmem_free(d_dest);

    CUDA_CHECK(hipStreamDestroy(cstrm));

    finalize_wrapper();

out:
    return 0;
}
