#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "bcast_latency.cubin"

#include "coll_test.h"
#define DATATYPE int64_t

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

#define CALL_BCAST(TYPENAME, TYPE, TG_PRE, THREADGROUP, THREAD_COMP, ELEM_COMP)                    \
    __global__ void test_##TYPENAME##_bcast_call_kern##THREADGROUP(                                \
        nvshmem_team_t team, TYPE *dest, const TYPE *source, int nelems, int mype, int PE_root,    \
        int iter) {                                                                                \
        int i;                                                                                     \
                                                                                                   \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP) && (nelems < ELEM_COMP)) {                  \
            for (i = 0; i < iter; i++) {                                                           \
                nvshmem##TG_PRE##_##TYPENAME##_broadcast##THREADGROUP(team, dest, source, nelems,  \
                                                                      PE_root);                    \
            }                                                                                      \
        }                                                                                          \
    }                                                                                              \
    void test_##TYPENAME##_bcast_call_kern##THREADGROUP##_cubin(                                   \
        int num_blocks, int num_tpb, hipStream_t stream, void **arglist) {                        \
        hipFunction_t test_cubin;                                                                     \
                                                                                                   \
        init_test_case_kernel(                                                                     \
            &test_cubin, NVSHMEMI_TEST_STRINGIFY(test_##TYPENAME##_bcast_call_kern##THREADGROUP)); \
        CU_CHECK(hipModuleLaunchCooperativeKernel(test_cubin, num_blocks, 1, 1, num_tpb, 1, 1, 0, stream, \
                                           arglist));                                              \
    }

#define CALL_BCAST_KERNEL(TYPENAME, THREADGROUP, BLOCKS, THREADS, ARG_LIST, STREAM)                \
    if (use_cubin) {                                                                               \
        test_##TYPENAME##_bcast_call_kern##THREADGROUP##_cubin(BLOCKS, THREADS, STREAM, ARG_LIST); \
    } else {                                                                                       \
        status = nvshmemx_collective_launch(                                                       \
            (const void *)test_##TYPENAME##_bcast_call_kern##THREADGROUP, BLOCKS, THREADS,         \
            ARG_LIST, 0, STREAM);                                                                  \
        if (status != NVSHMEMX_SUCCESS) {                                                          \
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);                      \
            exit(-1);                                                                              \
        }                                                                                          \
    }

CALL_BCAST(int32, int32_t, , , 1, 512);
CALL_BCAST(int64, int64_t, , , 1, 512);
CALL_BCAST(int32, int32_t, x, _warp, warpSize, 4096);
CALL_BCAST(int64, int64_t, x, _warp, warpSize, 4096);
CALL_BCAST(int32, int32_t, x, _block, INT_MAX, INT_MAX);
CALL_BCAST(int64, int64_t, x, _block, INT_MAX, INT_MAX);

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

int broadcast_calling_kernel(nvshmem_team_t team, void *dest, const void *source, int mype,
                             int PE_root, hipStream_t stream, void **h_tables) {
    int status = 0;
    int nvshm_test_num_tpb = threads_per_block;
    int num_blocks = 1;
    size_t num_elems = 1;
    size_t min_elems, max_elems;
    int i;
    int skip = warmup_iters;
    int iter = iters;
    int npes = nvshmem_n_pes();
    uint64_t *h_size_array = (uint64_t *)h_tables[0];
    double *h_thread_lat = (double *)h_tables[1];
    double *h_warp_lat = (double *)h_tables[2];
    double *h_block_lat = (double *)h_tables[3];
    float milliseconds;
    void *args_1[] = {&team, &dest, &source, &num_elems, &mype, &PE_root, &skip};
    void *args_2[] = {&team, &dest, &source, &num_elems, &mype, &PE_root, &iter};
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float *ms_d = (float *)nvshmem_malloc(sizeof(float));
    float *ms_sum_d = (float *)nvshmem_malloc(sizeof(float));

    nvshmem_barrier_all();
    min_elems = max(static_cast<size_t>(1), min_size / sizeof(int32_t));
    max_elems = max(static_cast<size_t>(1), max_size / sizeof(int32_t));
    i = 0;
    for (num_elems = min_elems; num_elems < 512; num_elems *= step_factor) {
        CALL_BCAST_KERNEL(int32, , num_blocks, nvshm_test_num_tpb, args_1, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        CALL_BCAST_KERNEL(int32, , num_blocks, nvshm_test_num_tpb, args_2, stream);

        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_thread_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = min_elems; num_elems < 4096; num_elems *= step_factor) {
        CALL_BCAST_KERNEL(int32, _warp, num_blocks, nvshm_test_num_tpb, args_1, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        CALL_BCAST_KERNEL(int32, _warp, num_blocks, nvshm_test_num_tpb, args_2, stream);

        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_warp_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = min_elems; num_elems <= max_elems; num_elems *= step_factor) {
        h_size_array[i] = calculate_collective_size("bcast", num_elems, sizeof(int32_t), npes);
        CALL_BCAST_KERNEL(int32, _block, num_blocks, nvshm_test_num_tpb, args_1, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        CALL_BCAST_KERNEL(int32, _block, num_blocks, nvshm_test_num_tpb, args_2, stream);

        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_block_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    if (!mype) {
        print_table_v1("bcast_device", "32-bit-thread", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_thread_lat, i);
        print_table_v1("bcast_device", "32-bit-warp", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_warp_lat, i);
        print_table_v1("bcast_device", "32-bit-block", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_block_lat, i);
    }

    min_elems = max(static_cast<size_t>(1), min_size / sizeof(int64_t));
    max_elems = max(static_cast<size_t>(1), max_size / sizeof(int64_t));
    i = 0;
    for (num_elems = min_elems; num_elems < 512; num_elems *= step_factor) {
        CALL_BCAST_KERNEL(int64, , num_blocks, nvshm_test_num_tpb, args_1, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        CALL_BCAST_KERNEL(int64, , num_blocks, nvshm_test_num_tpb, args_2, stream);

        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_thread_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = min_elems; num_elems < 4096; num_elems *= step_factor) {
        CALL_BCAST_KERNEL(int64, _warp, num_blocks, nvshm_test_num_tpb, args_1, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        CALL_BCAST_KERNEL(int64, _warp, num_blocks, nvshm_test_num_tpb, args_2, stream);

        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        if (!mype) {
            hipEventElapsedTime(&milliseconds, start, stop);
            h_warp_lat[i] = (milliseconds * 1000.0) / (float)iter;
        }
        i++;
        nvshmem_barrier_all();
    }

    i = 0;
    for (num_elems = min_elems; num_elems <= max_elems; num_elems *= step_factor) {
        h_size_array[i] = calculate_collective_size("bcast", num_elems, sizeof(int64_t), npes);
        CALL_BCAST_KERNEL(int64, _block, num_blocks, nvshm_test_num_tpb, args_1, stream);

        CUDA_CHECK(hipStreamSynchronize(stream));

        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        CALL_BCAST_KERNEL(int64, _block, num_blocks, nvshm_test_num_tpb, args_2, stream);

        hipEventRecord(stop, stream);
        CUDA_CHECK(hipStreamSynchronize(stream));

        hipEventElapsedTime(&milliseconds, start, stop);
        hipMemcpy(ms_d, &milliseconds, sizeof(float), hipMemcpyHostToDevice);
        nvshmem_float_sum_reduce(NVSHMEM_TEAM_WORLD, ms_sum_d, ms_d, 1);
        hipMemcpy(&milliseconds, ms_sum_d, sizeof(float), hipMemcpyDeviceToHost);
        if (!mype) {
            h_block_lat[i] =
                (milliseconds * 1000.0) / ((float)iter * nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD));
        }
        i++;
        nvshmem_barrier_all();
    }

    if (!mype) {
        print_table_v1("bcast_device", "64-bit-thread", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_thread_lat, i);
        print_table_v1("bcast_device", "64-bit-warp", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_warp_lat, i);
        print_table_v1("bcast_device", "64-bit-block", "size (Bytes)", "latency", "us", '-',
                       h_size_array, h_block_lat, i);
    }

    return status;
}

int main(int argc, char **argv) {
    int status = 0;
    int mype, array_size;

    read_args(argc, argv);
    size_t size = max_size * 2;
    size_t alloc_size;
    DATATYPE *buffer = NULL;
    DATATYPE *h_buffer = NULL;
    DATATYPE *d_source, *d_dest;
    DATATYPE *h_source, *h_dest;
    int root = 0;
    char size_string[100];
    hipStream_t cstrm;
    void **h_tables;

    array_size = max_size_log;
    if (use_mmap) {
        size = pad_up(size) * 2;
    }
    DEBUG_PRINT("symmetric size %lu\n", size);
    sprintf(size_string, "%lu", size);

    status = setenv("NVSHMEM_SYMMETRIC_SIZE", size_string, 1);
    if (status) {
        fprintf(stderr, "setenv failed \n");
        status = -1;
        goto out;
    }

    init_wrapper(&argc, &argv);
    alloc_tables(&h_tables, 4, array_size);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
    }

    mype = nvshmem_my_pe();
    CUDA_CHECK(hipStreamCreateWithFlags(&cstrm, hipStreamNonBlocking));

    DEBUG_PRINT("SHMEM: [%d of %d] hello shmem world! \n", mype,
                nvshmem_team_n_pes(NVSHMEM_TEAM_WORLD));

    alloc_size = max_size * 2;

    CUDA_CHECK(hipHostAlloc(&h_buffer, alloc_size, hipHostMallocDefault));
    h_source = (DATATYPE *)h_buffer;
    h_dest = (DATATYPE *)&h_source[max_size / sizeof(DATATYPE)];

    if (use_mmap) {
        buffer = (DATATYPE *)allocate_mmap_buffer(alloc_size, mem_handle_type, use_egm);
        DEBUG_PRINT("Allocating mmap buffer of size %zu\n", alloc_size);
    } else {
        buffer = (DATATYPE *)nvshmem_malloc(alloc_size);
        DEBUG_PRINT("Allocating nvshmem malloc buffer of size %zu\n", alloc_size);
    }

    if (!buffer) {
        fprintf(stderr, "buffer allocation failed \n");
        status = -1;
        goto out;
    }

    d_source = (DATATYPE *)buffer;
    d_dest = (DATATYPE *)&d_source[max_size / sizeof(DATATYPE)];

    for (int i = 0; i < max_size / sizeof(DATATYPE); i++) {
        h_source[i] = i;
    }

    CUDA_CHECK(hipMemcpyAsync(d_source, h_source, max_size, hipMemcpyHostToDevice, cstrm));
    CUDA_CHECK(hipMemcpyAsync(d_dest, h_dest, max_size, hipMemcpyHostToDevice, cstrm));

    broadcast_calling_kernel(NVSHMEM_TEAM_WORLD, d_dest, d_source, mype, root, cstrm, h_tables);

    CUDA_CHECK(hipMemcpyAsync(h_source, d_source, max_size, hipMemcpyDeviceToHost, cstrm));
    CUDA_CHECK(hipMemcpyAsync(h_dest, d_dest, max_size, hipMemcpyDeviceToHost, cstrm));

    nvshmem_barrier_all();

    CUDA_CHECK(hipHostFree(h_buffer));
    if (use_mmap) {
        free_mmap_buffer(buffer);
    } else {
        nvshmem_free(buffer);
    }

    CUDA_CHECK(hipStreamDestroy(cstrm));
    free_tables(h_tables, 4);
    finalize_wrapper();

out:
    return 0;
}
