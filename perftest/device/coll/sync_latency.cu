#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "sync_latency.cubin"

#include "coll_test.h"

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

#define SYNC_KERNEL(TG_PRE, THREADGROUP, THREAD_COMP, VARIANT, VARIANT_API, TEAM, TEAM_DELIM)      \
    void test_sync##TEAM_DELIM##TEAM##VARIANT##call_kernel##THREADGROUP##_cubin(                   \
        int num_blocks, int num_tpb, hipStream_t stream, void **arglist) {                        \
        hipFunction_t test_cubin;                                                                     \
                                                                                                   \
        init_test_case_kernel(                                                                     \
            &test_cubin, NVSHMEMI_TEST_STRINGIFY(                                                  \
                             test_sync##TEAM_DELIM##TEAM##VARIANT##call_kernel##THREADGROUP));     \
        CU_CHECK(hipModuleLaunchCooperativeKernel(test_cubin, num_blocks, 1, 1, num_tpb, 1, 1, 0, stream, \
                                           arglist));                                              \
    }                                                                                              \
                                                                                                   \
    __global__ void test_sync##TEAM_DELIM##TEAM##VARIANT##call_kernel##THREADGROUP(                \
        nvshmem_team_t team, int iter) {                                                           \
        int i;                                                                                     \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP)) {                                          \
            for (i = 0; i < iter; i++) {                                                           \
                nvshmem##TG_PRE##TEAM_DELIM##TEAM##_sync##VARIANT_API##THREADGROUP(TEAM);          \
            }                                                                                      \
        }                                                                                          \
    }

#define CALL_SYNC_KERNEL(THREADGROUP, BLOCKS, THREADS, ARG_LIST, STREAM, VARIANT)                  \
    if (use_cubin) {                                                                               \
        test_sync##VARIANT##call_kernel##THREADGROUP##_cubin(BLOCKS, THREADS, STREAM, ARG_LIST);   \
    } else {                                                                                       \
        status =                                                                                   \
            nvshmemx_collective_launch((const void *)test_sync##VARIANT##call_kernel##THREADGROUP, \
                                       BLOCKS, THREADS, ARG_LIST, 0, STREAM);                      \
        if (status != NVSHMEMX_SUCCESS) {                                                          \
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);                      \
            exit(-1);                                                                              \
        }                                                                                          \
    }

SYNC_KERNEL(, , 1, _, , team, _);
SYNC_KERNEL(x, _warp, warpSize, _, , team, _);
SYNC_KERNEL(x, _block, INT_MAX, _, , team, _);

SYNC_KERNEL(, , 1, _all_, _all, , );
SYNC_KERNEL(x, _warp, warpSize, _all_, _all, , );
SYNC_KERNEL(x, _block, INT_MAX, _all_, _all, , );

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

int sync_calling_kernel(nvshmem_team_t team, hipStream_t stream, int mype, void **h_tables) {
    int status = 0;
    int nvshm_test_num_tpb = threads_per_block;
    int skip = warmup_iters;
    int iter = iters;
    int num_blocks = 1;
    double *h_thread_lat = (double *)h_tables[0];
    double *h_warp_lat = (double *)h_tables[1];
    double *h_block_lat = (double *)h_tables[2];
    size_t size = 0;

    uint64_t tpb_size = (uint64_t)nvshm_test_num_tpb;

    void *sync_args_1[] = {&team, &skip};
    void *sync_args_2[] = {&team, &iter};
    void *sync_all_args_1[] = {&skip};
    void *sync_all_args_2[] = {&iter};
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nvshmem_barrier_all();
    CALL_SYNC_KERNEL(, num_blocks, nvshm_test_num_tpb, sync_args_1, stream, _team_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_SYNC_KERNEL(, num_blocks, nvshm_test_num_tpb, sync_args_2, stream, _team_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_thread_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_SYNC_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, sync_args_1, stream, _team_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_SYNC_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, sync_args_2, stream, _team_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_warp_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_SYNC_KERNEL(_block, num_blocks, nvshm_test_num_tpb, sync_args_1, stream, _team_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_SYNC_KERNEL(_block, num_blocks, nvshm_test_num_tpb, sync_args_2, stream, _team_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_block_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    if (!mype) {
        print_table_basic("sync_device", "thread", "threads per block", "latency", "us", '-',
                          &tpb_size, h_thread_lat, 1);
        print_table_basic("sync_device", "warp", "threads per block", "latency", "us", '-',
                          &tpb_size, h_warp_lat, 1);
        print_table_basic("sync_device", "block", "threads per block", "latency", "us", '-',
                          &tpb_size, h_block_lat, 1);
    }

    nvshmem_barrier_all();
    CALL_SYNC_KERNEL(, num_blocks, nvshm_test_num_tpb, sync_all_args_1, stream, _all_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();
    hipEventRecord(start, stream);
    CALL_SYNC_KERNEL(, num_blocks, nvshm_test_num_tpb, sync_all_args_2, stream, _all_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_thread_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_SYNC_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, sync_all_args_1, stream, _all_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_SYNC_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, sync_all_args_2, stream, _all_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_warp_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_SYNC_KERNEL(_block, num_blocks, nvshm_test_num_tpb, sync_all_args_1, stream, _all_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_SYNC_KERNEL(_block, num_blocks, nvshm_test_num_tpb, sync_all_args_2, stream, _all_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_block_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    if (!mype) {
        print_table_basic("sync_all_device", "thread", "threads per block", "latency", "us", '-',
                          &tpb_size, h_thread_lat, 1);
        print_table_basic("sync_all_device", "warp", "threads per block", "latency", "us", '-',
                          &tpb_size, h_warp_lat, 1);
        print_table_basic("sync_all_device", "block", "threads per block", "latency", "us", '-',
                          &tpb_size, h_block_lat, 1);
    }

    return status;
}

int main(int argc, char **argv) {
    int mype;
    hipStream_t cstrm;
    void **h_tables;

    read_args(argc, argv);
    init_wrapper(&argc, &argv);
    alloc_tables(&h_tables, 3, 1);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
    }

    mype = nvshmem_my_pe();
    CUDA_CHECK(hipStreamCreateWithFlags(&cstrm, hipStreamNonBlocking));

    sync_calling_kernel(NVSHMEM_TEAM_WORLD, cstrm, mype, h_tables);

    nvshmem_barrier_all();

    CUDA_CHECK(hipStreamDestroy(cstrm));
    free_tables(h_tables, 3);
    finalize_wrapper();

    return 0;
}
