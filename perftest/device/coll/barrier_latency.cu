#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "barrier_latency.cubin"

#include "coll_test.h"

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

#define BARRIER_KERNEL_WRAPPER(TG_PRE, THREADGROUP, THREAD_COMP, VARIANT, VARIANT_API)             \
    void test_barrier##VARIANT##call_kernel##VARIANT_API##THREADGROUP##_cubin(                     \
        int num_blocks, int num_tpb, hipStream_t stream, void **arglist) {                        \
        hipFunction_t test_cubin;                                                                     \
                                                                                                   \
        init_test_case_kernel(&test_cubin,                                                         \
                              NVSHMEMI_TEST_STRINGIFY(                                             \
                                  test_barrier##VARIANT##call_kernel##VARIANT_API##THREADGROUP));  \
        CU_CHECK(hipModuleLaunchCooperativeKernel(test_cubin, num_blocks, 1, 1, num_tpb, 1, 1, 0, stream, \
                                           arglist));                                              \
    }

#define BARRIER_KERNEL(TG_PRE, THREADGROUP, THREAD_COMP)                                   \
    __global__ void test_barrier_call_kernel##THREADGROUP(nvshmem_team_t team, int iter) { \
        int i;                                                                             \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP)) {                                  \
            for (i = 0; i < iter; i++) {                                                   \
                nvshmem##TG_PRE##_barrier##THREADGROUP(team);                              \
            }                                                                              \
        }                                                                                  \
    }                                                                                      \
                                                                                           \
    __global__ void test_barrier_all_call_kernel##THREADGROUP(int iter) {                  \
        int i;                                                                             \
        if (!blockIdx.x && (threadIdx.x < THREAD_COMP)) {                                  \
            for (i = 0; i < iter; i++) {                                                   \
                nvshmem##TG_PRE##_barrier_all##THREADGROUP();                              \
            }                                                                              \
        }                                                                                  \
    }

#define CALL_BARRIER_KERNEL(THREADGROUP, BLOCKS, THREADS, ARG_LIST, STREAM, VARIANT)        \
    if (use_cubin) {                                                                        \
        test_barrier##VARIANT##call_kernel##THREADGROUP##_cubin(BLOCKS, THREADS, STREAM,    \
                                                                ARG_LIST);                  \
    } else {                                                                                \
        status = nvshmemx_collective_launch(                                                \
            (const void *)test_barrier##VARIANT##call_kernel##THREADGROUP, BLOCKS, THREADS, \
            ARG_LIST, 0, STREAM);                                                           \
        if (status != NVSHMEMX_SUCCESS) {                                                   \
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);               \
            exit(-1);                                                                       \
        }                                                                                   \
    }

BARRIER_KERNEL(, , 1);
BARRIER_KERNEL(x, _warp, warpSize);
BARRIER_KERNEL(x, _block, INT_MAX);

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

BARRIER_KERNEL_WRAPPER(, , 1, _, );
BARRIER_KERNEL_WRAPPER(x, warp, warpSize, _, _);
BARRIER_KERNEL_WRAPPER(x, block, INT_MAX, _, _);
BARRIER_KERNEL_WRAPPER(, , 1, _all_, );
BARRIER_KERNEL_WRAPPER(x, warp, warpSize, _all_, _);
BARRIER_KERNEL_WRAPPER(x, block, INT_MAX, _all_, _);

int barrier_calling_kernel(nvshmem_team_t team, hipStream_t stream, int mype, void **h_tables) {
    int status = 0;
    int nvshm_test_num_tpb = threads_per_block;
    size_t skip = warmup_iters;
    size_t iter = iters;
    int num_blocks = 1;
    int npes = nvshmem_n_pes();
    double *h_thread_lat = (double *)h_tables[0];
    double *h_warp_lat = (double *)h_tables[1];
    double *h_block_lat = (double *)h_tables[2];
    uint64_t size = 0;
    uint64_t tpb_size = nvshm_test_num_tpb;
    void *barrier_args_1[] = {&team, &skip};
    void *barrier_args_2[] = {&team, &iter};
    void *barrier_all_args_1[] = {&skip};
    void *barrier_all_args_2[] = {&iter};
    float milliseconds;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nvshmem_barrier_all();
    CALL_BARRIER_KERNEL(, num_blocks, nvshm_test_num_tpb, barrier_args_1, stream, _)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_BARRIER_KERNEL(, num_blocks, nvshm_test_num_tpb, barrier_args_2, stream, _)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_thread_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_BARRIER_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, barrier_args_1, stream, _)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_BARRIER_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, barrier_args_2, stream, _)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_warp_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_BARRIER_KERNEL(_block, num_blocks, nvshm_test_num_tpb, barrier_args_1, stream, _)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_BARRIER_KERNEL(_block, num_blocks, nvshm_test_num_tpb, barrier_args_2, stream, _)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_block_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    if (!mype) {
        print_table_basic("barrier_device", "thread", "threads per block", "latency", "us", '-',
                          &tpb_size, h_thread_lat, 1);
        print_table_basic("barrier_device", "warp", "threads per block", "latency", "us", '-',
                          &tpb_size, h_warp_lat, 1);
        print_table_basic("barrier_device", "block", "threads per block", "latency", "us", '-',
                          &tpb_size, h_block_lat, 1);
    }

    nvshmem_barrier_all();
    CALL_BARRIER_KERNEL(, num_blocks, nvshm_test_num_tpb, barrier_all_args_1, stream, _all_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_BARRIER_KERNEL(, num_blocks, nvshm_test_num_tpb, barrier_all_args_2, stream, _all_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_thread_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_BARRIER_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, barrier_all_args_1, stream, _all_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_BARRIER_KERNEL(_warp, num_blocks, nvshm_test_num_tpb, barrier_all_args_2, stream, _all_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_warp_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    nvshmem_barrier_all();
    CALL_BARRIER_KERNEL(_block, num_blocks, nvshm_test_num_tpb, barrier_all_args_1, stream, _all_)

    CUDA_CHECK(hipStreamSynchronize(stream));

    nvshmem_barrier_all();

    hipEventRecord(start, stream);
    CALL_BARRIER_KERNEL(_block, num_blocks, nvshm_test_num_tpb, barrier_all_args_1, stream, _all_)

    hipEventRecord(stop, stream);
    CUDA_CHECK(hipStreamSynchronize(stream));

    if (!mype) {
        hipEventElapsedTime(&milliseconds, start, stop);
        h_block_lat[0] = (milliseconds * 1000.0) / (float)iter;
    }

    if (!mype) {
        print_table_basic("barrier_all_device", "thread", "threads per block", "latency", "us", '-',
                          &tpb_size, h_thread_lat, 1);
        print_table_basic("barrier_all_device", "warp", "threads per block", "latency", "us", '-',
                          &tpb_size, h_warp_lat, 1);
        print_table_basic("barrier_all_device", "block", "threads per block", "latency", "us", '-',
                          &tpb_size, h_block_lat, 1);
    }

    return status;
}

int main(int argc, char **argv) {
    int mype;
    hipStream_t cstrm;
    void **h_tables;

    read_args(argc, argv);
    init_wrapper(&argc, &argv);
    alloc_tables(&h_tables, 3, 1);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
    }

    mype = nvshmem_my_pe();
    CUDA_CHECK(hipStreamCreateWithFlags(&cstrm, hipStreamNonBlocking));

    barrier_calling_kernel(NVSHMEM_TEAM_WORLD, cstrm, mype, h_tables);

    nvshmem_barrier_all();

    CUDA_CHECK(hipStreamDestroy(cstrm));
    free_tables(h_tables, 3);
    finalize_wrapper();

    return 0;
}
