#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "utils.h"

#define UNROLL 2

template <typename T>
__device__ inline T call_nvshmem_g(T *rptr, int peer) {
    switch (sizeof(T)) {
        case 1:
            return nvshmem_uint8_g((uint8_t *)rptr, peer);
            break;
        case 2:
            return nvshmem_uint16_g((uint16_t *)rptr, peer);
            break;
        case 4:
            return nvshmem_uint32_g((uint32_t *)rptr, peer);
            break;
        case 8:
            return nvshmem_double_g((double *)rptr, peer);
            break;
        default:
            assert(0);
    }
    return (T)0;
}

template <typename T>
__global__ void bw(T *data_d, volatile unsigned int *counter_d, int len, int pe, int iter,
                   int stride) {
    int u, i, j, peer, tid, slice;
    unsigned int counter;
    int threads = gridDim.x * blockDim.x;
    tid = blockIdx.x * blockDim.x + threadIdx.x;

    peer = !pe;
    slice = UNROLL * threads * stride;

    // When stride > 1, each iteration requests less than len elements.
    // We increase the number of iterations to make up for that.
    for (i = 0; i < iter * stride; i++) {
        for (j = 0; j < len - slice; j += slice) {
            for (u = 0; u < UNROLL; ++u) {
                int idx = j + u * threads + tid * stride;
                *(data_d + idx) = call_nvshmem_g<T>(data_d + idx, peer);
            }
            __syncthreads(); /* This is required for performance over PCIe. PCIe has a P2P mailbox
                                protocol that has a window of 64KB for device BAR addresses. Not
                                synchronizing
                                across threads will lead to jumping in and out of the 64K window */
        }

        for (u = 0; u < UNROLL; ++u) {
            int idx = j + u * threads + tid * stride;
            if (idx < len) *(data_d + idx) = call_nvshmem_g<T>(data_d + idx, peer);
        }

        // synchronizing across blocks
        __syncthreads();

        if (!threadIdx.x) {
            __threadfence(); /* To ensure that the data received through shmem_g is
                                visible across the gpu */
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }

        __syncthreads();
    }

    // synchronizing across blocks
    __syncthreads();

    if (!threadIdx.x) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }
}

void call_bw(int blocks, int threads, void *data_d, unsigned int *counter_d, size_t size,
             NVSHMEM_DATATYPE_T dt, int mype, int iter, int stride) {
    switch (dt) {
        case NVSHMEM_INT:
            bw<int><<<blocks, threads>>>((int *)data_d, counter_d, size / sizeof(uint8_t), mype,
                                         iter, stride);
            break;
        case NVSHMEM_LONG:
            bw<long><<<blocks, threads>>>((long *)data_d, counter_d, size / sizeof(uint16_t), mype,
                                          iter, stride);
            break;
        case NVSHMEM_LONGLONG:
            bw<long long><<<blocks, threads>>>((long long *)data_d, counter_d,
                                               size / sizeof(uint32_t), mype, iter, stride);
            break;
        case NVSHMEM_ULONGLONG:
            bw<unsigned long long><<<blocks, threads>>>((unsigned long long *)data_d, counter_d,
                                                        size / sizeof(double), mype, iter, stride);
            break;
        case NVSHMEM_FLOAT:
            bw<float><<<blocks, threads>>>((float *)data_d, counter_d, size / sizeof(double), mype,
                                           iter, stride);
            break;
        case NVSHMEM_DOUBLE:
            bw<double><<<blocks, threads>>>((double *)data_d, counter_d, size / sizeof(double),
                                            mype, iter, stride);
            break;
        case NVSHMEM_UINT:
            bw<unsigned int><<<blocks, threads>>>((unsigned int *)data_d, counter_d,
                                                  size / sizeof(double), mype, iter, stride);
            break;
        case NVSHMEM_INT32:
            bw<int32_t><<<blocks, threads>>>((int32_t *)data_d, counter_d, size / sizeof(double),
                                             mype, iter, stride);
            break;
        case NVSHMEM_UINT32:
            bw<uint32_t><<<blocks, threads>>>((uint32_t *)data_d, counter_d, size / sizeof(double),
                                              mype, iter, stride);
            break;
        case NVSHMEM_INT64:
            bw<int64_t><<<blocks, threads>>>((int64_t *)data_d, counter_d, size / sizeof(double),
                                             mype, iter, stride);
            break;
        case NVSHMEM_UINT64:
            bw<uint64_t><<<blocks, threads>>>((uint64_t *)data_d, counter_d, size / sizeof(double),
                                              mype, iter, stride);
            break;
        case NVSHMEM_FP16:
            bw<half><<<blocks, threads>>>((half *)data_d, counter_d, size / sizeof(double), mype,
                                          iter, stride);
            break;
#if CUDA_VERSION >= 12020
        case NVSHMEM_BF16:
            bw<__hip_bfloat16><<<blocks, threads>>>((__hip_bfloat16 *)data_d, counter_d,
                                                   size / sizeof(double), mype, iter, stride);
            break;
#endif
        default:
            fprintf(stderr, "element=%d is not supported \n", dt);
            exit(-EINVAL);
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    void *data_d = NULL;
    unsigned int *counter_d;

    read_args(argc, argv);
    int max_blocks = num_blocks, max_threads = threads_per_block;
    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_bw;
    double *h_msgrate;
    bool report_msgrate = false;

    int iter = iters;
    int skip = warmup_iters;
    int element_size = datatype.size;

    float milliseconds;
    hipEvent_t start, stop;

    init_wrapper(&argc, &argv);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 3, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_bw = (double *)h_tables[1];
    h_msgrate = (double *)h_tables[2];

    if (use_mmap) {
        data_d = (void *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (void *)nvshmem_malloc(max_size);
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

    CUDA_CHECK(hipDeviceSynchronize());

    size_t size;
    i = 0;
    if (mype == 0) {
        for (size = min_size; size <= max_size; size *= step_factor) {
            int blocks = max_blocks, threads = max_threads;
            h_size_arr[i] = size;
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));
            call_bw(blocks, threads, data_d, counter_d, size, datatype.type, mype, skip, stride);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            hipEventRecord(start);
            call_bw(blocks, threads, data_d, counter_d, size, datatype.type, mype, iter, stride);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            hipEventElapsedTime(&milliseconds, start, stop);
            h_bw[i] = size / (milliseconds * (B_TO_GB / (iter * MS_TO_S)));
            h_msgrate[i] = (double)(size / element_size) * iter / (milliseconds * MS_TO_S);
            nvshmem_barrier_all();
            i++;
        }
    } else {
        for (size = min_size; size <= max_size; size *= step_factor) {
            nvshmem_barrier_all();
        }
    }

    if (mype == 0) {
        print_table_basic("shmem_g_bw", "None", "size (Bytes)", "BW", "GB/sec", '+', h_size_arr,
                          h_bw, i);
        if (report_msgrate)
            print_table_basic("shmem_g_bw", "None", "size (Bytes)", "msgrate", "MMPS", '+',
                              h_size_arr, h_msgrate, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    free_tables(h_tables, 3);
    finalize_wrapper();

    return 0;
}
