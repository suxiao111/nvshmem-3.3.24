#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "shmem_put_ping_pong_latency.cubin"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

__global__ void ping_pong(int *data_d, uint64_t *flag_d, int len, int pe, int iter) {
    int i, peer;
    peer = !pe;

    for (i = 0; i < iter; i++) {
        if (pe) {
            nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));

            nvshmem_int_put_nbi(data_d, data_d, len, peer);

            nvshmem_fence();

            nvshmemx_signal_op(flag_d, i + 1, NVSHMEM_SIGNAL_SET, peer);
        } else {
            nvshmem_int_put_nbi(data_d, data_d, len, peer);

            nvshmem_fence();

            nvshmemx_signal_op(flag_d, i + 1, NVSHMEM_SIGNAL_SET, peer);

            nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));
        }
    }
    nvshmem_quiet();
}

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

void test_ping_pong(void **arglist, hipFunction_t kernel, hipStream_t stream) {
    int status;
    if (use_cubin) {
        CU_CHECK(hipModuleLaunchCooperativeKernel(kernel, 1, 1, 1, 1, 1, 1, 0, stream, arglist));
    } else {
        status = nvshmemx_collective_launch((const void *)ping_pong, 1, 1, arglist, 0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    uint64_t *flag_d = NULL;
    int *data_d = NULL;
    hipStream_t stream;

    read_args(argc, argv);
    int iter = iters;
    int skip = warmup_iters;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    float milliseconds;
    hipEvent_t start, stop;
    hipFunction_t test_cubin = NULL;

    init_wrapper(&argc, &argv);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
        init_test_case_kernel(&test_cubin, "ping_pong");
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    if (use_mmap) {
        data_d = (int *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        flag_d = (uint64_t *)allocate_mmap_buffer(sizeof(uint64_t), mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (int *)nvshmem_malloc(max_size);
        flag_d = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
        CUDA_CHECK(hipMemset(flag_d, 0, sizeof(int)));
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];

    CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    if (mype == 0) {
        printf("Note: This test measures full round-trip latency\n");
    }

    i = 0;
    for (size_t size = min_size; size <= max_size; size *= step_factor) {
        int nelems, status = 0;
        nelems = size / sizeof(int);
        h_size_arr[i] = size;
        void *args_1[] = {&data_d, &flag_d, &nelems, &mype, &skip};
        void *args_2[] = {&data_d, &flag_d, &nelems, &mype, &iter};

        if (use_egm) {
            memset(flag_d, 0, sizeof(uint64_t));
        } else {
            CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        }
        CUDA_CHECK(hipDeviceSynchronize());
        nvshmem_barrier_all();

        test_ping_pong(args_1, test_cubin, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        CUDA_CHECK(hipDeviceSynchronize());
        if (use_egm) {
            memset(flag_d, 0, sizeof(uint64_t));
        } else {
            CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        }
        nvshmem_barrier_all();

        hipEventRecord(start, stream);
        test_ping_pong(args_2, test_cubin, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop, stream);

        /* give latency in us */
        CUDA_CHECK(hipEventSynchronize(stop));
        hipEventElapsedTime(&milliseconds, start, stop);
        h_lat[i] = (milliseconds * 1000) / iter;
        nvshmem_barrier_all();
        i++;
    }

    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) {
        print_table_basic("shmem_put_ping_lat", "None", "size (Bytes)", "latency", "us", '-',
                          h_size_arr, h_lat, i);
    }
finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    if (flag_d) {
        if (use_mmap) {
            free_mmap_buffer(flag_d);
        } else {
            nvshmem_free(flag_d);
        }
    }
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
