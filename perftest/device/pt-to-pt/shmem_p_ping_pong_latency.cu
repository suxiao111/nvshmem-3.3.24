#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define CUMODULE_NAME "shmem_p_ping_pong_latency.cubin"
#define UNROLL 8

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

__global__ void ping_pong(int *data_d, uint64_t *flag_d, int len, int pe, int iter) {
    int i, j, tid, peer;

    peer = !pe;
    tid = threadIdx.x;

    for (i = 0; i < iter; i++) {
        if (pe) {
            if (!tid) {
                nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));
            }
            __syncthreads();

            for (j = tid; j < len; j += blockDim.x) {
                nvshmem_int_p(data_d + j, *(data_d + j), peer);
            }
            __syncthreads();

            if (!tid) {
                nvshmem_fence();
                nvshmemx_signal_op(flag_d, (i + 1), NVSHMEM_SIGNAL_SET, peer);
            }
            __syncthreads();
        } else {
            for (j = tid; j < len; j += blockDim.x) {
                nvshmem_int_p(data_d + j, *(data_d + j), peer);
            }
            __syncthreads();

            if (!tid) {
                nvshmem_fence();
                nvshmemx_signal_op(flag_d, (i + 1), NVSHMEM_SIGNAL_SET, peer);
            }
            __syncthreads();

            if (!tid) {
                nvshmem_uint64_wait_until(flag_d, NVSHMEM_CMP_EQ, (i + 1));
            }
            __syncthreads();
        }
    }

    if (!tid) nvshmem_quiet();
}

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

void test_ping_pong(void **arglist, hipFunction_t kernel, hipStream_t stream) {
    int status;
    if (use_cubin) {
        CU_CHECK(hipModuleLaunchCooperativeKernel(kernel, 1, 1, 1, threads_per_block, 1, 1, 0, stream,
                                           arglist));
    } else {
        status = nvshmemx_collective_launch((const void *)ping_pong, 1, threads_per_block, arglist,
                                            0, stream);
        if (status != NVSHMEMX_SUCCESS) {
            fprintf(stderr, "shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    uint64_t *flag_d = NULL;
    int *data_d = NULL;

    sleep(10);

    read_args(argc, argv);
    int iter = iters;
    int skip = warmup_iters;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    float milliseconds;
    hipEvent_t start, stop;
    hipFunction_t test_cubin = NULL;

    init_wrapper(&argc, &argv);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
        init_test_case_kernel(&test_cubin, "ping_pong");
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];
    if (use_mmap) {
        data_d = (int *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        flag_d = (uint64_t *)allocate_mmap_buffer(sizeof(uint64_t), mem_handle_type, use_egm);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (int *)nvshmem_malloc(max_size);
        flag_d = (uint64_t *)nvshmem_malloc(sizeof(uint64_t));
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) {
        printf("Note: This test measures full round-trip latency\n");
    }

    i = 0;
    for (size_t size = min_size; size <= max_size; size *= step_factor) {
        int nelems, status = 0;
        nelems = size / sizeof(int);
        h_size_arr[i] = size;
        void *args_1[5] = {&data_d, &flag_d, &nelems, &mype, &skip};
        void *args_2[5] = {&data_d, &flag_d, &nelems, &mype, &iter};

        if (use_egm) {
            memset(flag_d, 0, sizeof(uint64_t));
        } else {
            CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        }
        CUDA_CHECK(hipDeviceSynchronize());
        nvshmem_barrier_all();
        test_ping_pong(args_1, test_cubin, 0);
        CUDA_CHECK(hipDeviceSynchronize());
        if (use_egm) {
            memset(flag_d, 0, sizeof(uint64_t));
        } else {
            CUDA_CHECK(hipMemset(flag_d, 0, sizeof(uint64_t)));
        }

        hipEventRecord(start);
        test_ping_pong(args_2, test_cubin, 0);
        if (status != NVSHMEMX_SUCCESS) {
            printf("shmemx_collective_launch failed %d \n", status);
            exit(-1);
        }
        hipEventRecord(stop);

        /* give latency in us */
        CUDA_CHECK(hipEventSynchronize(stop));
        hipEventElapsedTime(&milliseconds, start, stop);
        h_lat[i] = (milliseconds * 1000) / iter;
        nvshmem_barrier_all();
        i++;
    }

    if (mype == 0) {
        print_table_basic("shmem_ping_pong_lat", "None", "size (Bytes)", "latency", "us", '-',
                          h_size_arr, h_lat, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    if (flag_d) {
        if (use_mmap) {
            free_mmap_buffer(flag_d);
        } else {
            nvshmem_free(flag_d);
        }
    }
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
