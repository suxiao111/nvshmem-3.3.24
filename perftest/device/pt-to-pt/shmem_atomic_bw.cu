#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION   All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto   Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "shmem_atomic_bw.cubin"

#include "atomic_bw_common.h"

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

DEFINE_ATOMIC_BW_FN_NO_ARG(inc);
DEFINE_ATOMIC_BW_FN_NO_ARG(fetch_inc);

DEFINE_ATOMIC_BW_FN_ONE_ARG(add, 1);
DEFINE_ATOMIC_BW_FN_ONE_ARG(fetch_add, 1);

DEFINE_ATOMIC_BW_FN_ONE_ARG(and, (*(data_d + idx) << (i + 1)));
DEFINE_ATOMIC_BW_FN_ONE_ARG(fetch_and, (*(data_d + idx) << (i + 1)));

DEFINE_ATOMIC_BW_FN_ONE_ARG(or, (*(data_d + idx) << i));
DEFINE_ATOMIC_BW_FN_ONE_ARG(fetch_or, (*(data_d + idx) << i));

DEFINE_ATOMIC_BW_FN_ONE_ARG(xor, 1);
DEFINE_ATOMIC_BW_FN_ONE_ARG(fetch_xor, 1);

DEFINE_ATOMIC_BW_FN_ONE_ARG(swap, i + 1);
DEFINE_ATOMIC_BW_FN_ONE_ARG(set, i + 1);

DEFINE_ATOMIC_BW_FN_TWO_ARG(compare_swap, i, i + 1);

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

int main(int argc, char *argv[]) {
    int mype, npes;
    int size;
    int nelems;
    uint64_t *data_d = NULL;
    uint64_t set_value;
    unsigned int *counter_d;
    read_args(argc, argv);

    int max_blocks = num_blocks, max_threads = threads_per_block;
    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_bw;
    char perf_table_name[30];

    int iter = iters;
    int skip = warmup_iters;

    float milliseconds;
    hipEvent_t start, stop;

    void *args_skip[] = {(void *)&data_d, (void *)&counter_d, (void *)&(nelems), (void *)&mype,
                         (void *)&skip};
    void *args_iter[] = {(void *)&data_d, (void *)&counter_d, (void *)&(nelems), (void *)&mype,
                         (void *)&iter};

    init_wrapper(&argc, &argv);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes   \n");
        goto finalize;
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_bw = (double *)h_tables[1];

    if (use_mmap) {
        data_d = (uint64_t *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (uint64_t *)nvshmem_malloc(max_size);
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

    CUDA_CHECK(hipDeviceSynchronize());

    strncpy(perf_table_name, ("shmem_atomic_" + test_amo.name).c_str(), 30);

    i = 0;
    if (mype == 0) {
        for (size = min_size; size <= max_size; size *= step_factor) {
            int blocks = max_blocks, threads = max_threads;
            nelems = size / sizeof(uint64_t);
            h_size_arr[i] = size;
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            /* Do warmup round for NIC cache. */
            switch (test_amo.type) {
                case AMO_INC: {
                    CALL_ATOMIC_BW_KERNEL(inc, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_SET: {
                    CALL_ATOMIC_BW_KERNEL(set, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_ADD: {
                    CALL_ATOMIC_BW_KERNEL(add, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_AND: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    CALL_ATOMIC_BW_KERNEL(and, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_OR: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    CALL_ATOMIC_BW_KERNEL(or, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_XOR: {
                    set_value = 1;
                    for (size_t j = 0; j < size / sizeof(uint64_t); j++) {
                        hipMemcpy((data_d + j), &set_value, sizeof(uint64_t),
                                   hipMemcpyHostToDevice);
                    }
                    CALL_ATOMIC_BW_KERNEL(xor, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_FETCH_INC: {
                    CALL_ATOMIC_BW_KERNEL(fetch_inc, blocks, threads, data_d, counter_d, nelems,
                                          mype, skip, args_skip)
                    break;
                }
                case AMO_FETCH_ADD: {
                    CALL_ATOMIC_BW_KERNEL(fetch_add, blocks, threads, data_d, counter_d, nelems,
                                          mype, skip, args_skip)
                    break;
                }
                case AMO_FETCH_AND: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    CALL_ATOMIC_BW_KERNEL(fetch_and, blocks, threads, data_d, counter_d, nelems,
                                          mype, skip, args_skip)
                    break;
                }
                case AMO_FETCH_OR: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    CALL_ATOMIC_BW_KERNEL(fetch_or, blocks, threads, data_d, counter_d, nelems,
                                          mype, skip, args_skip)
                    break;
                }
                case AMO_FETCH_XOR: {
                    for (size_t j = 0; j < nelems; j++) {
                        hipMemcpy((data_d + j), &set_value, sizeof(uint64_t),
                                   hipMemcpyHostToDevice);
                    }
                    CALL_ATOMIC_BW_KERNEL(fetch_xor, blocks, threads, data_d, counter_d, nelems,
                                          mype, skip, args_skip)
                    break;
                }
                case AMO_SWAP: {
                    CALL_ATOMIC_BW_KERNEL(swap, blocks, threads, data_d, counter_d, nelems, mype,
                                          skip, args_skip)
                    break;
                }
                case AMO_COMPARE_SWAP: {
                    CALL_ATOMIC_BW_KERNEL(compare_swap, blocks, threads, data_d, counter_d, nelems,
                                          mype, skip, args_skip)
                    break;
                }
                default: {
                    /* Should be unreachable */
                    fprintf(stderr, "Error, unsupported Atomic op %d.\n", test_amo.type);
                    goto finalize;
                }
            }
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
            nvshmem_barrier_all();

            /* reset values in code. */
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));
            switch (test_amo.type) {
                case AMO_AND: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    break;
                }
                case AMO_OR: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    break;
                }
                case AMO_XOR: {
                    set_value = 1;
                    for (size_t j = 0; j < size / sizeof(uint64_t); j++) {
                        hipMemcpy((data_d + j), &set_value, sizeof(uint64_t),
                                   hipMemcpyHostToDevice);
                    }
                    break;
                }
                case AMO_FETCH_AND: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    break;
                }
                case AMO_FETCH_OR: {
                    if (use_egm) {
                        memset(data_d, 0xFF, size);
                    } else {
                        CUDA_CHECK(hipMemset(data_d, 0xFF, size));
                    }
                    break;
                }
                case AMO_FETCH_XOR: {
                    for (size_t j = 0; j < size / sizeof(uint64_t); j++) {
                        hipMemcpy((data_d + j), &set_value, sizeof(uint64_t),
                                   hipMemcpyHostToDevice);
                    }
                    break;
                }
                default: { break; }
            }
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
            nvshmem_barrier_all();

            hipEventRecord(start);
            switch (test_amo.type) {
                case AMO_INC: {
                    CALL_ATOMIC_BW_KERNEL(inc, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_SET: {
                    CALL_ATOMIC_BW_KERNEL(set, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_ADD: {
                    CALL_ATOMIC_BW_KERNEL(add, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_AND: {
                    CALL_ATOMIC_BW_KERNEL(and, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_OR: {
                    CALL_ATOMIC_BW_KERNEL(or, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_XOR: {
                    CALL_ATOMIC_BW_KERNEL(xor, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_FETCH_INC: {
                    CALL_ATOMIC_BW_KERNEL(fetch_inc, blocks, threads, data_d, counter_d, nelems,
                                          mype, iter, args_iter)
                    break;
                }
                case AMO_FETCH_ADD: {
                    CALL_ATOMIC_BW_KERNEL(fetch_add, blocks, threads, data_d, counter_d, nelems,
                                          mype, iter, args_iter)
                    break;
                }
                case AMO_FETCH_AND: {
                    CALL_ATOMIC_BW_KERNEL(fetch_and, blocks, threads, data_d, counter_d, nelems,
                                          mype, iter, args_iter)
                    break;
                }
                case AMO_FETCH_OR: {
                    CALL_ATOMIC_BW_KERNEL(fetch_or, blocks, threads, data_d, counter_d, nelems,
                                          mype, iter, args_iter)
                    break;
                }
                case AMO_FETCH_XOR: {
                    CALL_ATOMIC_BW_KERNEL(fetch_xor, blocks, threads, data_d, counter_d, nelems,
                                          mype, iter, args_iter)
                    break;
                }
                case AMO_SWAP: {
                    CALL_ATOMIC_BW_KERNEL(swap, blocks, threads, data_d, counter_d, nelems, mype,
                                          iter, args_iter)
                    break;
                }
                case AMO_COMPARE_SWAP: {
                    CALL_ATOMIC_BW_KERNEL(compare_swap, blocks, threads, data_d, counter_d, nelems,
                                          mype, iter, args_iter)
                    break;
                }
                default: {
                    /* Should be unreachable */
                    fprintf(stderr, "Error, unsupported Atomic op %d.\n", test_amo.type);
                    goto finalize;
                }
            }
            hipEventRecord(stop);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));
            hipEventElapsedTime(&milliseconds, start, stop);

            h_bw[i] = size / (milliseconds * (B_TO_GB / (iter * MS_TO_S)));
            nvshmem_barrier_all();
            i++;
        }
    } else {
        for (size = min_size; size <= max_size; size *= step_factor) {
            nvshmem_barrier_all();
            nvshmem_barrier_all();
            nvshmem_barrier_all();
        }
    }

    if (mype == 0) {
        print_table_basic(perf_table_name, "None", "size (Bytes)", "BW", "GB/sec", '+', h_size_arr,
                          h_bw, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
