#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "shmem_p_latency.cubin"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

__global__ void p_latency(int *data_d, int len, int pe, int iter) {
    int i, j, tid, peer;

    peer = !pe;
    tid = threadIdx.x;

    for (i = 0; i < iter; i++) {
        for (j = tid; j < len; j += blockDim.x) {
            nvshmem_int_p(data_d + j, *(data_d + j), peer);
        }
        __syncthreads();
        if (!tid) {
            nvshmem_quiet();
        }
        __syncthreads();
    }
}

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

void test_p(int *data_d, int len, int pe, int iter, hipFunction_t kernel) {
    if (use_cubin) {
        void *arglist[] = {(void *)&data_d, (void *)&len, (void *)&pe, (void *)&iter};
        CU_CHECK(hipModuleLaunchKernel(kernel, 1, 1, 1, threads_per_block, 1, 1, 0, NULL, arglist, NULL));
    } else {
        p_latency<<<1, threads_per_block>>>(data_d, len, pe, iter);
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    int *data_d = NULL;

    read_args(argc, argv);

    int iter = iters;
    int skip = warmup_iters;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    float milliseconds;
    hipEvent_t start, stop;
    hipFunction_t test_cubin = NULL;

    init_wrapper(&argc, &argv);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
        init_test_case_kernel(&test_cubin, "p_latency");
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];

    if (use_mmap) {
        data_d = (int *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
        if (!data_d) {
            fprintf(stderr, "buffer allocation failed \n");
            goto finalize;
        }
    } else {
        data_d = (int *)nvshmem_malloc(max_size);
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        if (!data_d) {
            fprintf(stderr, "buffer allocation failed \n");
            goto finalize;
        }
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());

    i = 0;
    for (size_t size = min_size; size <= max_size; size *= step_factor) {
        if (!mype) {
            int nelems;
            h_size_arr[i] = size;
            nelems = size / sizeof(int);

            test_p(data_d, nelems, mype, skip, test_cubin);
            hipEventRecord(start);
            test_p(data_d, nelems, mype, iter, test_cubin);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            hipEventElapsedTime(&milliseconds, start, stop);
            /* give latency in us */
            h_lat[i] = (milliseconds * 1000) / iter;
            i++;
        }

        nvshmem_barrier_all();
    }

    if (mype == 0) {
        print_table_basic("shmem_p_latency", "None", "size (Bytes)", "latency", "us", '-',
                          h_size_arr, h_lat, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
