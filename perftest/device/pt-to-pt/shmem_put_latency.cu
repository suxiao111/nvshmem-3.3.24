#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#define CUMODULE_NAME "shmem_put_latency.cubin"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include "utils.h"

#define THREADS_PER_WARP 32

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
extern "C" {
#endif

__global__ void latency_kern(int *data_d, int len, int pe, int iter) {
    int i, peer;

    peer = !pe;

    for (i = 0; i < iter; i++) {
        nvshmem_int_put_nbi(data_d, data_d, len, peer);
        nvshmem_quiet();
    }
}

#define LATENCY_THREADGROUP(group)                                                 \
    __global__ void latency_kern_##group(int *data_d, int len, int pe, int iter) { \
        int i, tid, peer;                                                          \
                                                                                   \
        peer = !pe;                                                                \
        tid = threadIdx.x;                                                         \
                                                                                   \
        for (i = 0; i < iter; i++) {                                               \
            nvshmemx_int_put_##group(data_d, data_d, len, peer);                   \
                                                                                   \
            __syncthreads();                                                       \
            if (!tid) nvshmem_quiet();                                             \
            __syncthreads();                                                       \
        }                                                                          \
    }

LATENCY_THREADGROUP(warp)
LATENCY_THREADGROUP(block)

#if defined __cplusplus || defined NVSHMEM_HOSTLIB_ONLY
}
#endif

#define DEFINE_TEST_LATENCY(TG)                                                               \
                                                                                              \
    void test_latency##TG(int *data_d, int len, int pe, int iter, hipFunction_t kernel,          \
                          int threads) {                                                      \
        if (use_cubin) {                                                                      \
            void *arglist[] = {(void *)&data_d, (void *)&len, (void *)&pe, (void *)&iter};    \
            CU_CHECK(hipModuleLaunchKernel(kernel, 1, 1, 1, threads, 1, 1, 0, NULL, arglist, NULL)); \
        } else {                                                                              \
            latency_kern##TG<<<1, threads>>>(data_d, len, pe, iter);                          \
        }                                                                                     \
    }

DEFINE_TEST_LATENCY()
DEFINE_TEST_LATENCY(_warp)
DEFINE_TEST_LATENCY(_block)

int main(int argc, char *argv[]) {
    int mype, npes, size;
    int *data_d = NULL;

    read_args(argc, argv);
    int iter = iters;
    int skip = warmup_iters;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_lat;

    float milliseconds;
    hipEvent_t start, stop;
    hipFunction_t test_cubin = NULL;
    hipFunction_t test_cubin_warp = NULL;
    hipFunction_t test_cubin_block = NULL;

    init_wrapper(&argc, &argv);

    if (use_cubin) {
        init_cumodule(CUMODULE_NAME);
        init_test_case_kernel(&test_cubin, "latency_kern");
        init_test_case_kernel(&test_cubin_warp, "latency_kern_warp");
        init_test_case_kernel(&test_cubin_block, "latency_kern_block");
    }

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    if (use_mmap) {
        data_d = (int *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (int *)nvshmem_malloc(max_size);
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_lat = (double *)h_tables[1];

    nvshmem_barrier_all();

    CUDA_CHECK(hipDeviceSynchronize());

    i = 0;
    for (size = min_size; size <= max_size; size *= step_factor) {
        if (!mype) {
            int nelems;
            h_size_arr[i] = size;
            nelems = size / sizeof(int);

            test_latency(data_d, nelems, mype, skip, test_cubin, 1);
            hipEventRecord(start);
            test_latency(data_d, nelems, mype, iter, test_cubin, 1);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            /* give latency in us */
            hipEventElapsedTime(&milliseconds, start, stop);
            h_lat[i] = (milliseconds * 1000) / iter;
            i++;
        }

        nvshmem_barrier_all();
    }

    if (mype == 0) {
        print_table_basic("shmem_put_latency", "Thread", "size (Bytes)", "latency", "us", '-',
                          h_size_arr, h_lat, i);
    }

    i = 0;
    for (size = min_size; size <= max_size; size *= step_factor) {
        if (!mype) {
            int nelems;
            h_size_arr[i] = size;
            nelems = size / sizeof(int);

            test_latency_warp(data_d, nelems, mype, skip, test_cubin_warp, THREADS_PER_WARP);
            hipEventRecord(start);
            test_latency_warp(data_d, nelems, mype, iter, test_cubin_warp, THREADS_PER_WARP);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            /* give latency in us */
            hipEventElapsedTime(&milliseconds, start, stop);
            h_lat[i] = (milliseconds * 1000) / iter;
            i++;
        }

        nvshmem_barrier_all();
    }

    if (mype == 0) {
        print_table_basic("shmem_put_latency", "Warp", "size (Bytes)", "latency", "us", '-',
                          h_size_arr, h_lat, i);
    }

    i = 0;
    for (size = min_size; size <= max_size; size *= step_factor) {
        if (!mype) {
            int nelems;
            h_size_arr[i] = size;
            nelems = size / sizeof(int);

            test_latency_block(data_d, nelems, mype, skip, test_cubin_block, threads_per_block);
            hipEventRecord(start);
            test_latency_block(data_d, nelems, mype, iter, test_cubin_block, threads_per_block);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            /* give latency in us */
            hipEventElapsedTime(&milliseconds, start, stop);
            h_lat[i] = (milliseconds * 1000) / iter;
            i++;
        }

        nvshmem_barrier_all();
    }

    if (mype == 0) {
        print_table_basic("shmem_put_latency", "Block", "size (Bytes)", "latency", "us", '-',
                          h_size_arr, h_lat, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    free_tables(h_tables, 2);

    finalize_wrapper();

    return 0;
}
