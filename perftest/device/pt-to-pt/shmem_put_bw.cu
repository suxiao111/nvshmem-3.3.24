#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "utils.h"

__global__ void bw(double *data_d, volatile unsigned int *counter_d, int len, int pe, int iter) {
    int i, peer;
    unsigned int counter;
    int tid = (threadIdx.x * blockDim.y * blockDim.z + threadIdx.y * blockDim.z + threadIdx.z);
    int bid = blockIdx.x;
    int nblocks = gridDim.x;

    peer = !pe;
    for (i = 0; i < iter; i++) {
        nvshmemx_double_put_nbi_block(data_d + (bid * (len / nblocks)),
                                      data_d + (bid * (len / nblocks)), len / nblocks, peer);

        // synchronizing across blocks
        __syncthreads();
        if (!tid) {
            __threadfence();
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }
        __syncthreads();
    }

    // synchronize and call nvshme_quiet
    __syncthreads();
    if (!tid) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            nvshmem_quiet();
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }
    __syncthreads();
}

int main(int argc, char *argv[]) {
    int mype, npes;
    double *data_d = NULL;
    unsigned int *counter_d;

    read_args(argc, argv);
    int max_blocks = num_blocks, max_threads = threads_per_block;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_bw = NULL, *h_bw_total = NULL;
    double *d_bw = NULL, *d_bw_sum = NULL;

    int iter = iters;
    int skip = warmup_iters;

    float milliseconds;
    hipEvent_t start, stop;

    init_wrapper(&argc, &argv);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_bw = (double *)h_tables[1];

    if (bidirectional) {
        h_bw_total = (double *)malloc(sizeof(double) * array_size);

        if (!h_bw_total) {
            fprintf(stderr, "Error: Unable to malloc on the host.\n");
            exit(1);
        }

        memset(h_bw_total, 0, sizeof(double) * array_size);

        d_bw = (double *)nvshmem_malloc(sizeof(double));
        d_bw_sum = (double *)nvshmem_malloc(sizeof(double));
    }

    if (use_mmap) {
        data_d = (double *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (double *)nvshmem_malloc(max_size);
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

    CUDA_CHECK(hipDeviceSynchronize());

    if (bidirectional || mype == 0) {
        i = 0;
        for (size_t size = min_size; size <= max_size; size *= step_factor) {
            h_size_arr[i] = size;
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));
            bw<<<max_blocks, max_threads>>>(data_d, counter_d, size / sizeof(double), mype, skip);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            hipEventRecord(start);
            bw<<<max_blocks, max_threads>>>(data_d, counter_d, size / sizeof(double), mype, iter);
            hipEventRecord(stop);

            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            hipEventElapsedTime(&milliseconds, start, stop);
            h_bw[i] = size / (milliseconds * (B_TO_GB / (iter * MS_TO_S)));
            nvshmem_barrier_all();

            /* Sum all h_bw of each PE for bidirectional mode. */
            if (bidirectional) {
                CUDA_CHECK(hipMemcpy(d_bw, &h_bw[i], sizeof(double), hipMemcpyDefault));
                nvshmem_double_sum_reduce(NVSHMEM_TEAM_WORLD, d_bw_sum, d_bw, 1);
                CUDA_CHECK(hipMemcpy(&h_bw_total[i], d_bw_sum, sizeof(double), hipMemcpyDefault));
            }

            i++;
        }
    } else {
        for (int size = min_size; size <= max_size; size *= step_factor) {
            nvshmem_barrier_all();
        }
    }

    if (mype == 0) {
        double *p_h_bw_tmp = bidirectional ? h_bw_total : h_bw;
        const char *const test_name = bidirectional ? "shmem_put_bw_bidi" : "shmem_put_bw_uni";
        print_table_basic(test_name, "None", "size (Bytes)", "BW", "GB/sec", '+', h_size_arr,
                          p_h_bw_tmp, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }

    if (d_bw) nvshmem_free(d_bw);
    if (d_bw_sum) nvshmem_free(d_bw_sum);

    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
