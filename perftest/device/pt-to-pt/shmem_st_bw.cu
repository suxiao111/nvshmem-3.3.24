#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <getopt.h>
#include "utils.h"

#define UNROLL 2

__global__ void bw(double *data_d, double *remote_d, volatile unsigned int *counter_d, int len,
                   int pe, int iter) {
    int u, i, j, tid, slice;
    unsigned int counter;
    int threads = gridDim.x * blockDim.x;
    tid = blockIdx.x * blockDim.x + threadIdx.x;

    slice = UNROLL * threads;

    for (i = 0; i < iter; i++) {
        for (j = 0; j < len - slice; j += slice) {
            for (u = 0; u < UNROLL; ++u) {
                int idx = j + u * threads + tid;
                *(remote_d + idx) = *(data_d + idx);
            }
            __syncthreads();
        }

        for (u = 0; u < UNROLL; ++u) {
            int idx = j + u * threads + tid;
            if (idx < len) *(remote_d + idx) = *(data_d + idx);
        }

        // synchronizing across blocks
        __syncthreads();

        if (!threadIdx.x) {
            __threadfence();
            counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
            if (counter == (gridDim.x * (i + 1) - 1)) {
                *(counter_d + 1) += 1;
            }
            while (*(counter_d + 1) != i + 1)
                ;
        }

        __syncthreads();
    }

    // synchronizing across blocks
    __syncthreads();

    if (!threadIdx.x) {
        __threadfence();
        counter = atomicInc((unsigned int *)counter_d, UINT_MAX);
        if (counter == (gridDim.x * (i + 1) - 1)) {
            nvshmem_quiet();
            *(counter_d + 1) += 1;
        }
        while (*(counter_d + 1) != i + 1)
            ;
    }
}

int main(int argc, char *argv[]) {
    int mype, npes;
    double *data_d = NULL, *remote_d;
    unsigned int *counter_d;

    read_args(argc, argv);
    int max_blocks = num_blocks, max_threads = threads_per_block;

    int iter = iters;
    int skip = warmup_iters;

    int array_size, i;
    void **h_tables;
    uint64_t *h_size_arr;
    double *h_bw;

    float milliseconds;
    hipEvent_t start, stop;

    init_wrapper(&argc, &argv);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) {
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }

    array_size = max_size_log;
    alloc_tables(&h_tables, 2, array_size);
    h_size_arr = (uint64_t *)h_tables[0];
    h_bw = (double *)h_tables[1];

    if (use_mmap) {
        data_d = (double *)allocate_mmap_buffer(max_size, mem_handle_type, use_egm, true);
        DEBUG_PRINT("Allocated mmap buffer\n");
    } else {
        data_d = (double *)nvshmem_malloc(max_size);
        DEBUG_PRINT("Allocated nvshmem malloc buffer\n");
        CUDA_CHECK(hipMemset(data_d, 0, max_size));
    }

    remote_d = (double *)nvshmem_ptr((void *)data_d, !mype);
    if (remote_d == NULL) {
        fprintf(stderr, "peer memory not accessible for LD/ST \n");
        goto finalize;
    }

    CUDA_CHECK(hipMalloc((void **)&counter_d, sizeof(unsigned int) * 2));
    CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) {
        printf("Size(Bytes) \t\t BWGB/sec\n");
        fflush(stdout);
    }

    int size;
    i = 0;
    if (mype == 0) {
        for (size = min_size; size <= max_size; size *= step_factor) {
            int blocks = max_blocks, threads = max_threads;
            h_size_arr[i] = size;

            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            bw<<<blocks, threads>>>(data_d, remote_d, counter_d, size / sizeof(double), mype, skip);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipDeviceSynchronize());
            CUDA_CHECK(hipMemset(counter_d, 0, sizeof(unsigned int) * 2));

            hipEventRecord(start);
            bw<<<blocks, threads>>>(data_d, remote_d, counter_d, size / sizeof(double), mype, iter);
            hipEventRecord(stop);
            CUDA_CHECK(hipGetLastError());
            CUDA_CHECK(hipEventSynchronize(stop));

            hipEventElapsedTime(&milliseconds, start, stop);
            h_bw[i] = size / (milliseconds * (B_TO_GB / (iter * MS_TO_S)));
            nvshmem_barrier_all();
            i++;
        }
    } else {
        for (size = min_size; size <= max_size; size *= step_factor) {
            nvshmem_barrier_all();
        }
    }

    if (mype == 0) {
        print_table_basic("shmem_st_bw", "None", "size (Bytes)", "BW", "GB/sec", '+', h_size_arr,
                          h_bw, i);
    }

finalize:

    if (data_d) {
        if (use_mmap) {
            free_mmap_buffer(data_d);
        } else {
            nvshmem_free(data_d);
        }
    }
    free_tables(h_tables, 2);
    finalize_wrapper();

    return 0;
}
