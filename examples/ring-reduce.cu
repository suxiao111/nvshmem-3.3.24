#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

/* This example performs an allreduce operation using ring algorithm when
   GPUs are connected via remote interconect like IB/RoCE/EFA, etc.
   It does ring reduce followed by ring broadcast. We use single threaded put_signal API
   as single thread is sufficient for remote transfers. The example is expected
   to be performant only when GPUs are connected via remote interconnect. */

#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <nvshmem.h>
#include <nvshmemx.h>
#include <unistd.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#undef CUDA_CHECK
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] cuda failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

/* atol() + optional scaled suffix recognition: 1K, 2M, 3G, 1T */
static inline int atol_scaled(const char *str, size_t *out) {
    int scale, n;
    double p = -1.0;
    char f;
    n = sscanf(str, "%lf%c", &p, &f);

    if (n == 2) {
        switch (f) {
            case 'k':
            case 'K':
                scale = 10;
                break;
            case 'm':
            case 'M':
                scale = 20;
                break;
            case 'g':
            case 'G':
                scale = 30;
                break;
            case 't':
            case 'T':
                scale = 40;
                break;
            default:
                return 1;
        }
    } else if (p < 0) {
        return 1;
    } else
        scale = 0;

    *out = (size_t)ceil(p * (1lu << scale));
    return 0;
}

size_t min_size = 1024 * 1024 * 32;
size_t max_size = min_size * 16;
size_t num_blocks = 32;
size_t threads_per_block = 512;
size_t iters = 4;
size_t warmup_iters = 1;
size_t step_factor = 2;
size_t chunk_size = 262144;

// perform Allreduce using ring
__global__ void ring_reduce(int *dst, const int *src, size_t nreduce, uint64_t *signal,
                            size_t chunk_size) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    int thread_id = threadIdx.x;
    int num_threads = blockDim.x;
    int num_blocks = gridDim.x;
    int block_idx = blockIdx.x;
    size_t elems_per_block = nreduce / num_blocks;

    // Change src, dst, nreduce, signal to what this block is going to process
    // Each CTA will work independently
    if (elems_per_block * (blockIdx.x + 1) > nreduce) return;
    src = src + block_idx * elems_per_block;
    dst = dst + block_idx * elems_per_block;
    nreduce = elems_per_block;
    signal = signal + block_idx;

    size_t chunk_elems = chunk_size / sizeof(int);
    size_t num_chunks = nreduce / chunk_elems;

    // reduce phase
    for (size_t chunk = 0; chunk < num_chunks; chunk++) {
        if (mype != 0) {
            if (thread_id == 0) nvshmem_signal_wait_until(signal, NVSHMEM_CMP_GE, chunk + 1);

            __syncthreads();
            for (size_t i = thread_id; i < chunk_elems; i += num_threads) {
                dst[i] = dst[i] + src[i];
            }
            __syncthreads();
        }
        if (thread_id == 0)
            nvshmem_int_put_signal_nbi(dst, (mype == 0) ? src : dst, chunk_elems, signal, 1,
                                       NVSHMEM_SIGNAL_ADD, peer);
        src = src + chunk_elems;
        dst = dst + chunk_elems;
    }

    // Broadcast phase
    dst = dst - num_chunks * chunk_elems;
    if (thread_id == 0) {
        for (size_t chunk = 0; chunk < num_chunks; chunk++) {
            if (mype < npes - 1) {  // Last pe already has the final result
                nvshmem_signal_wait_until(signal, NVSHMEM_CMP_GE,
                                          (mype == 0) ? chunk + 1 : num_chunks + chunk + 1);
            }
            if (mype < npes - 2)
                nvshmem_int_put_signal_nbi(dst, dst, chunk_elems, signal, 1, NVSHMEM_SIGNAL_ADD,
                                           peer);
            dst = dst + chunk_elems;
        }
        *signal = 0;  // reset for next iteration
    }
}

int main(int argc, char **argv) {
    int c;
    while ((c = getopt(argc, argv, "b:e:f:n:w:c:t:m:")) != -1) {
        switch (c) {
            case 'b':
                atol_scaled(optarg, &min_size);
                break;
            case 'e':
                atol_scaled(optarg, &max_size);
                break;
            case 'f':
                atol_scaled(optarg, &step_factor);
                break;
            case 'n':
                atol_scaled(optarg, &iters);
                break;
            case 'w':
                atol_scaled(optarg, &warmup_iters);
                break;
            case 'c':
                atol_scaled(optarg, &num_blocks);
                break;
            case 't':
                atol_scaled(optarg, &threads_per_block);
                break;
            case 'm':
                atol_scaled(optarg, &chunk_size);
                break;
            case '?':
                if (optopt == 'c')
                    fprintf(stderr, "Option -%c requires an argument.\n", optopt);
                else if (isprint(optopt))
                    fprintf(stderr, "Unknown option `-%c'.\n", optopt);
                else
                    fprintf(stderr, "Unknown option character `\\x%x'.\n", optopt);
                return 1;
            default:
                abort();
        }
    }
    size_t min_ints = min_size / sizeof(int);
    assert(min_ints % num_blocks == 0);

    nvshmem_init();

    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    hipStream_t stream;
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));

    size_t max_ints = max_size / sizeof(int);
    int *dst = (int *)nvshmem_malloc(max_size);
    int *src = (int *)nvshmem_malloc(max_size);
    int *data_h = (int *)malloc(max_size);
    uint64_t *signal = (uint64_t *)nvshmem_calloc(num_blocks, sizeof(uint64_t));
    dim3 gridDim(num_blocks), blockDim(threads_per_block);

    for (size_t i = 0; i < max_ints; i++) data_h[i] = i;

    CUDA_CHECK(hipMemcpyAsync(src, data_h, max_size, hipMemcpyHostToDevice, stream));
    nvshmemx_barrier_all_on_stream(stream);

    for (size_t size = min_size; size <= max_size; size *= step_factor) {
        size_t num_ints = size / sizeof(int);
        void *args[] = {&dst, &src, &num_ints, &signal, &chunk_size};

        // do warmup
        for (size_t i = 0; i < warmup_iters; i++) {
            nvshmemx_collective_launch((const void *)ring_reduce, gridDim, blockDim, args, 0,
                                       stream);
            nvshmemx_barrier_all_on_stream(stream);
        }
        CUDA_CHECK(hipStreamSynchronize(stream));

        // main loop
        CUDA_CHECK(hipEventRecord(start, stream));
        for (size_t i = 0; i < iters; i++) {
            nvshmemx_collective_launch((const void *)ring_reduce, gridDim, blockDim, args, 0,
                                       stream);
            nvshmemx_barrier_all_on_stream(stream);
        }
        CUDA_CHECK(hipEventRecord(stop, stream));

        CUDA_CHECK(hipStreamSynchronize(stream));
        if (!mype) {
            float ms;
            CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
            printf("%zuB \t %fms\n", size, ms / iters);
        }

        // validate output
        CUDA_CHECK(hipMemcpy(data_h, dst, size, hipMemcpyDeviceToHost));
        for (size_t i = 0; i < num_ints; i++) {
            if (data_h[i] != (int)i * npes)
                printf("PE %d error, data[%zu] = %d expected data[%zu] = %d\n", mype, i, data_h[i],
                       i, (int)i * npes);
        }
    }

    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    nvshmem_free(dst);
    nvshmem_free(src);
    nvshmem_free(signal);
    free(data_h);

    nvshmem_finalize();
    return 0;
}
