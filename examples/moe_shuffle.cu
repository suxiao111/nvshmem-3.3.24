#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * NVIDIA CORPORATION and its licensors retain all intellectual property
 * and proprietary rights in and to this software, related documentation
 * and any modifications thereto.  Any use, reproduction, disclosure or
 * distribution of this software and related documentation without an express
 * license agreement from NVIDIA CORPORATION is strictly prohibited.
 *
 * See COPYRIGHT.txt for license information
 */

#include <algorithm>
#include <chrono>
#include <functional>
#include <getopt.h>
#include <iomanip>
#include <iostream>
#include <stdio.h>
#include <vector>

#include "nvshmem.h"
#include "nvshmemx.h"

/* Types */
typedef enum {
    MOE_BARE_TWO_STEP_ALLPUSH = 0x0,
    MOE_COLL_LAUNCH_TWO_STEP_ALLPUSH = 0x1,
    MOE_BARE_TWO_STEP_BY_PEER = 0x2,
    MOE_COLL_LAUNCH_TWO_STEP_BY_PEER = 0x3,
    MOE_ONE_STEP_LAUNCH_MIN = 0x4,
    MOE_BARE_ONE_STEP_ALLPUSH = 0x4,
    MOE_COLL_LAUNCH_ONE_STEP_ALLPUSH = 0x5,
    MOE_BARE_ONE_STEP_BY_PEER = 0x6,
    MOE_COLL_LAUNCH_ONE_STEP_BY_PEER = 0x7
} moe_comms_profile_t;

/* Constants */
#define CUDA_CHECK(stmt)                                                          \
    do {                                                                          \
        hipError_t result = (stmt);                                              \
        if (hipSuccess != result) {                                              \
            fprintf(stderr, "[%s:%d] CUDA failed with %s \n", __FILE__, __LINE__, \
                    hipGetErrorString(result));                                  \
            exit(-1);                                                             \
        }                                                                         \
    } while (0)

constexpr int num_experts = 16;

/* Start of initialization helpers */
void set_comms_profile(int *profile, bool do_by_peer, bool do_one_step, bool do_collective_launch) {
    if (do_by_peer) {
        *profile = MOE_BARE_TWO_STEP_BY_PEER;
    }

    *profile += do_collective_launch;
    *profile += do_one_step ? MOE_ONE_STEP_LAUNCH_MIN : 0;
}

int validate_input_params(int *num_blocks, int num_baseline_blocks, int threads_per_block,
                          int num_rows, int num_elems) {
    int rc = 0;
    int num_warps_per_block = threads_per_block / 32;
    int num_elems_per_row = num_rows / num_elems;

    *num_blocks = std::min(num_baseline_blocks, num_rows);  // best for 16 ranks
    if (*num_blocks < num_baseline_blocks) {
        std::cout << "number of blocks requested (" << num_baseline_blocks << ")"
                  << "is greater than the number of rows. (" << num_rows
                  << ") decreasing block count to " << num_blocks << "\n";
    }

    /* for bench simplicity */
    if (num_elems % threads_per_block) {
        std::cout << "num_elems (" << num_elems << ") is not evenly divisible by "
                  << "threads_per_block (" << threads_per_block << "). Cannot continue\n";
        rc = -1;
    }

    if (threads_per_block % 32) {
        std::cout << "num_threads_per_block (" << threads_per_block
                  << ") is not evenly divisible by num_threads_per_warp"
                  << " (32). Cannot continue\n";
        rc = -1;
    }

    if (num_elems_per_row % num_warps_per_block) {
        std::cout << "num_elems_per_row (" << num_elems_per_row
                  << ") is not evenly divisible by num_warps_per_block (" << num_warps_per_block
                  << "). Cannot continue.\n";
    }

    /* for bench simplicity */
    if (num_rows % *num_blocks) {
        std::cout << "num_rows (" << num_rows << ") is not evenly divisible by num_blocks ("
                  << *num_blocks << "). Cannot continue\n";
        rc = -1;
    }

    return rc;
}

void usage(void) {
    std::cout
        << "USAGE -b [number of blocks] -t [threads per block] -c -h -l -o -p -r -v\n"
        << "-c use hipMalloc to instantiate expert to row mappings rather than malloc\n"
        << "-h display this help message\n"
        << "-l use nvshmemx_collective_launch to launch the MoE alltoallv kernel\n"
        << "-o perform the allgather of offsets in the same kernel as the alltoallv\n"
        << "-p use a peer based communication pattern rather than writing to all peers at once\n"
        << "-r randomize the selection of experts per rows, creating an uneven alltoall pattern\n"
        << "-v display verbose output about the selected parameters\n";
}

int setup_test_parameters(int argc, char **argv, int *num_blocks, int *threads_per_block,
                          bool *use_cuda_malloc, bool *randomize_output, bool *warp_specialized,
                          moe_comms_profile_t *profile, int num_rows, int num_elems) {
    int num_baseline_blocks = 96;
    int rc = 0;
    bool do_by_peer = 0;
    bool do_one_step = 0;
    bool do_collective_launch = 0;
    bool verbose = 0;

    while (1) {
        int c;
        c = getopt(argc, argv, "b:t:chloprvw");
        if (c == -1) break;

        switch (c) {
            case 'b':
                num_baseline_blocks = strtol(optarg, NULL, 0);
                break;
            case 't':
                *threads_per_block = strtol(optarg, NULL, 0);
                break;
            case 'c':
                *use_cuda_malloc = true;
                break;
            case 'l':
                do_collective_launch = true;
                break;
            case 'o':
                do_one_step = true;
                break;
            case 'p':
                do_by_peer = true;
                break;
            case 'r':
                *randomize_output = true;
                break;
            case 'v':
                verbose = true;
            case 'w':
                *warp_specialized = true;
                break;
            default:
                std::cout << "Received unknown argument: -" << c
                          << " Displaying help and exiting\n";
            case 'h':
                usage();
                rc = -1;
                goto finalize;
        }
    }

    set_comms_profile((int *)profile, do_by_peer, do_one_step, do_collective_launch);
    rc = validate_input_params(num_blocks, num_baseline_blocks, *threads_per_block, num_rows,
                               num_elems);
    if (rc) {
        goto finalize;
    }

    if (verbose) {
        std::cout
            << "performing moe comm pattern simulation with the following parameters: \n"
            << "number of blocks:                  " << *num_blocks << "\n"
            << "number of threads per block:       " << *threads_per_block << "\n"
            << "comms pattern:                     "
            << (do_by_peer ? "by peer                   " : "one shot                  ") << "\n"
            << "expert selection pattern:          "
            << (*randomize_output ? "randomized                " : "static                    ")
            << "\n"
            << "expert map allocation strategy:    "
            << (*use_cuda_malloc ? "hipMalloc               " : "malloc                   ")
            << "\n"
            << "operation compostion:              "
            << (do_one_step ? "single kernel            " : "split kernels            ") << "\n"
            << "alltoall launch strategy:          "
            << (do_collective_launch ? "nvshmem collective launch" : "direct launch            ")
            << "\n"
            << "warp specialization strategy:      "
            << (*warp_specialized ? "warp APIs                " : "block APIs               ")
            << "\n";
    }

finalize:
    return rc;
}
/* End of initialization helpers */

/* Start of offset exchange code */
static __forceinline__ __device__ void _exchange_offsets(int64_t *local_expert_counts,
                                                         int64_t *symmetric_expert_counts,
                                                         int64_t *accumulated_expert_positions,
                                                         int npes) {
    const int src_rank = threadIdx.x / npes;
    const int expert = threadIdx.x % npes;
    const int num_experts_per_rank = num_experts / npes;
    const int base = npes * num_experts_per_rank * threadIdx.x;

    int64_t prev = 0;

    // get counts from every node for each expert
    if (threadIdx.x < npes * num_experts) {
        local_expert_counts[threadIdx.x] =
            nvshmem_int64_g((int64_t *)symmetric_expert_counts + expert, src_rank);
    }
    __syncthreads();

    if (threadIdx.x < npes) {
#pragma unroll 4
        for (int i = 0; i < npes * num_experts_per_rank; ++i) {
            prev += local_expert_counts[base + i];
            accumulated_expert_positions[base + i] = prev;
        }
    }
}

__global__ void exchange_offsets(int64_t *expert_counts, int64_t *expert_pos_out, int npes) {
    extern __shared__ int64_t expert_count_in[];
    _exchange_offsets(expert_count_in, expert_counts, expert_pos_out, npes);
}

/* End of offset exchange code */

/* Start of alltoall code */
/* Start of allpush code */
template <bool WARP_SCOPED>
static __forceinline__ __device__ void _token_shuffle_allpush(
    float *send_data, float *recv_data, int *src_rows, int *src_experts, int64_t *expert_offsets,
    int k, int num_rows, int mype, int npes, int num_copies_per_block, int hidden_dim,
    int64_t *expert_pos_out) {
    const int num_experts_per_rank = num_experts / npes;
    const int num_warps_per_block = (blockDim.x * blockDim.y * blockDim.z) / 32;
    const int num_elems_per_warp = hidden_dim / num_warps_per_block;
    const int my_warp_idx = threadIdx.x / 32;

    int block_offset = blockIdx.x * num_copies_per_block;

    for (int i = 0; i < num_copies_per_block; ++i) {
        if (block_offset >= num_rows) {
            return;
        }
        /*
         * Copies one token (row)
         * All threads in the block call API with same arguments
         */
        auto src_row = src_rows[block_offset] % (num_rows / k);
        auto expert = src_experts[block_offset];
        auto peer = expert / num_experts_per_rank;
        bool first_expert_in_rank = expert % num_experts_per_rank == 0;

        // expert position on peer for this rank on the destination rank
        auto expert_start =
            (mype == 0 && first_expert_in_rank) ? 0 : expert_pos_out[mype + expert * npes - 1];
        // relative position in expert
        auto pos_in_expert = block_offset - (expert > 0 ? expert_offsets[expert - 1] : 0);

        /*
        if (threadIdx.x == 0) {
            printf("%3d %3d %3d %3ld %3ld \n", peer, expert, mype, expert_start, pos_in_expert);
        }
        */
        if (WARP_SCOPED) {
            nvshmemx_float_put_nbi_warp(
                recv_data + (expert_start + pos_in_expert) * hidden_dim +
                    my_warp_idx * num_elems_per_warp,
                send_data + src_row * hidden_dim + my_warp_idx * num_elems_per_warp,
                num_elems_per_warp, peer);
        } else {
            nvshmemx_float_put_nbi_block(recv_data + (expert_start + pos_in_expert) * hidden_dim,
                                         send_data + src_row * hidden_dim, hidden_dim, peer);
        }

        block_offset += 1;
    }
}

template <bool WARP_SCOPED>
__global__ void token_shuffle_two_step_allpush(float *send_data, float *recv_data, int *src_rows,
                                               int *src_experts, int64_t *expert_offsets, int k,
                                               int num_rows, int mype, int npes, int hidden_dim,
                                               int num_copies_per_block, int64_t *expert_pos_out) {
    _token_shuffle_allpush<WARP_SCOPED>(send_data, recv_data, src_rows, src_experts, expert_offsets,
                                        k, num_rows, mype, npes, num_copies_per_block, hidden_dim,
                                        expert_pos_out);
}

template <bool WARP_SCOPED>
__global__ void token_shuffle_one_step_allpush(float *send_data, float *recv_data, int *src_rows,
                                               int *src_experts, int64_t *expert_offsets, int k,
                                               int num_rows, int mype, int npes, int hidden_dim,
                                               int num_copies_per_block, int64_t *expert_counts) {
    extern __shared__ int64_t expert_count_positions[];

    _exchange_offsets(expert_count_positions, (expert_counts + num_experts * blockIdx.x),
                      &expert_count_positions[npes * num_experts], npes);
    __syncthreads();
    _token_shuffle_allpush<WARP_SCOPED>(send_data, recv_data, src_rows, src_experts, expert_offsets,
                                        k, num_rows, mype, npes, num_copies_per_block, hidden_dim,
                                        &expert_count_positions[npes * num_experts]);
}
/* End of allpush code */

/* Start of py peer code */
template <bool WARP_SCOPED>
static __forceinline__ __device__ void _token_shuffle_by_peer(
    float *send_data, float *recv_data, int *src_rows, int *src_experts, int64_t *expert_offsets,
    int k, int num_rows, int mype, int npes, int hidden_dim, int64_t *expert_positions) {
    const int num_experts_per_rank = num_experts / npes;
    const int num_warps_per_block = (blockDim.x * blockDim.y * blockDim.z) / 32;
    const int num_elems_per_warp = hidden_dim / num_warps_per_block;
    const int my_warp_idx = threadIdx.x / 32;

    int block_offset = blockIdx.x;
    int num_blocks = gridDim.x * gridDim.y * gridDim.z;
    int rows_per_expert = num_rows / num_experts;
    int true_block_offset = rows_per_expert * mype * num_experts_per_rank + block_offset;
    int rounded_block_offset = true_block_offset % num_rows;

    for (; block_offset < num_rows; block_offset += num_blocks) {
        if (block_offset >= num_rows) {
            return;
        }
        /*
         * Copies one token (row)
         * All threads in the block call API with same arguments
         */
        auto src_row = src_rows[rounded_block_offset] % (num_rows / k);
        auto expert = src_experts[rounded_block_offset];
        auto peer = expert / num_experts_per_rank;
        bool first_expert_in_rank = expert % num_experts_per_rank == 0;

        // expert position on peer for this rank on the destination rank
        auto expert_start =
            (mype == 0 && first_expert_in_rank) ? 0 : expert_positions[mype + expert * npes - 1];
        // relative position in expert
        auto pos_in_expert = rounded_block_offset - (expert > 0 ? expert_offsets[expert - 1] : 0);

        /* if (threadIdx.x == 0) {
            printf("mype: %3d block: %3d peer: %3d expert: %3d expert_start: %3ld pos_in_expert:
        %3ld \n", mype, blockIdx.x, peer, expert, expert_start, pos_in_expert);
        } */

        if (WARP_SCOPED) {
            nvshmemx_float_put_nbi_warp(
                recv_data + (expert_start + pos_in_expert) * hidden_dim +
                    my_warp_idx * num_elems_per_warp,
                send_data + src_row * hidden_dim + my_warp_idx * num_elems_per_warp,
                num_elems_per_warp, peer);
        } else {
            nvshmemx_float_put_nbi_block(recv_data + (expert_start + pos_in_expert) * hidden_dim,
                                         send_data + src_row * hidden_dim, hidden_dim, peer);
        }
        true_block_offset += num_blocks;
        rounded_block_offset = true_block_offset % num_rows;
    }
}

template <bool WARP_SCOPED>
__global__ void token_shuffle_two_step_by_peer(float *send_data, float *recv_data, int *src_rows,
                                               int *src_experts, int64_t *expert_offsets, int k,
                                               int num_rows, int mype, int npes, int hidden_dim,
                                               int64_t *expert_pos_out) {
    _token_shuffle_by_peer<WARP_SCOPED>(send_data, recv_data, src_rows, src_experts, expert_offsets,
                                        k, num_rows, mype, npes, hidden_dim, expert_pos_out);
}

template <bool WARP_SCOPED>
__global__ void token_shuffle_one_step_by_peer(float *send_data, float *recv_data, int *src_rows,
                                               int *src_experts, int64_t *expert_offsets, int k,
                                               int num_rows, int mype, int npes, int hidden_dim,
                                               int64_t *expert_counts) {
    extern __shared__ int64_t expert_count_positions[];

    _exchange_offsets(expert_count_positions, (expert_counts + num_experts * blockIdx.x),
                      &expert_count_positions[npes * num_experts], npes);
    __syncthreads();
    _token_shuffle_by_peer<WARP_SCOPED>(send_data, recv_data, src_rows, src_experts, expert_offsets,
                                        k, num_rows, mype, npes, hidden_dim,
                                        &expert_count_positions[npes * num_experts]);
}
/* End of py peer code */
/* End of alltoall code */

/* helper kernel for setting initial values */
__global__ void set_counts(float *pointer, int base, int hidden_dim) {
    int row = blockIdx.x;
    for (int col = threadIdx.x; col < hidden_dim; col += hidden_dim / blockDim.x) {
        pointer[row * hidden_dim + col] = row + base * 1000 + (float)col / hidden_dim;
    }
}

int main(int argc, char *argv[]) {
    constexpr int iterations = 50;     // number of iterations to time
    constexpr int num_src_rows = 384;  // local rows to start
    constexpr int expert_factor = 2;   // max input elements - unsafe

    int k = 2;                              // top K
    int num_rows = num_src_rows * k;        // total rows sent
    int hidden_dim = 16384 / 4;             // because we're sending floats
    int num_elems = num_rows * hidden_dim;  // total elements sent

    std::vector<std::pair<int, int>> expertForSrcRow(num_rows);
    std::vector<int64_t> expertCount(num_experts);
    std::vector<int64_t> totalExpertCount(num_experts);
    std::vector<int64_t> expertOffsetsCpu(num_experts);
    std::vector<int> expandedSrcRow(num_rows);
    std::vector<int> expertForExpandedSrcRow(num_rows);
    std::function<void()> run;

    void *cooperative_launch_args[13];
    int64_t *expert_counts_gpu_tmp;
    int64_t *expert_counts_gpu;
    int64_t *expert_offsets_gpu;
    int64_t *expert_pos_out_gpu;
    float *send_data, *recv_data;
    int *expandedSrcRow_gpu;
    int *expertForExpandedSrcRow_gpu;

    size_t offset_exchange_shared_memory;
    size_t one_step_shared_memory;
    float milliseconds;
    int gridsize;
    int mype, mype_node, npes;
    int rc;
    int nvshmem_rc;
    int num_blocks;
    int num_copies_per_block;

    hipStream_t stream;
    hipEvent_t start, stop;

    /* command line controlled variables */
    int threads_per_block = 1024;
    bool use_cuda_malloc = 0;
    bool randomize_output = 0;
    bool warp_specialized = 0;
    moe_comms_profile_t comms_profile = MOE_BARE_TWO_STEP_ALLPUSH;

    rc = setup_test_parameters(argc, argv, &num_blocks, &threads_per_block, &use_cuda_malloc,
                               &randomize_output, &warp_specialized, &comms_profile, num_rows,
                               num_elems);

    if (rc) {
        return -1;
    }

    num_copies_per_block = num_rows / num_blocks;

    nvshmem_init();
    npes = nvshmem_n_pes();
    mype = nvshmem_my_pe();
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);

    if (randomize_output) {
        srand(mype * 1000);
    }

    /* kernel shared memory calculations */
    offset_exchange_shared_memory = sizeof(int64_t) * npes * num_experts;
    one_step_shared_memory = sizeof(int64_t) * npes * num_experts * 2;

    /* CUDA state initialization */
    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    /* symmetric memory allocation */
    expert_counts_gpu = (int64_t *)nvshmem_malloc(sizeof(int64_t) * num_experts * num_blocks);
    expert_offsets_gpu = (int64_t *)nvshmem_malloc(sizeof(int64_t) * num_experts);
    send_data = (float *)nvshmem_malloc(sizeof(float) * num_src_rows * hidden_dim);
    recv_data = (float *)nvshmem_malloc(sizeof(float) * num_rows * hidden_dim * expert_factor);

    /* user buffer allocation*/
    CUDA_CHECK(hipMalloc(&expert_pos_out_gpu, sizeof(int64_t) * npes * num_experts * num_blocks));
    if (use_cuda_malloc) {
        CUDA_CHECK(hipMalloc(&expandedSrcRow_gpu, sizeof(int) * num_rows));
        CUDA_CHECK(hipMalloc(&expertForExpandedSrcRow_gpu, sizeof(int) * num_rows));
    } else {
        expandedSrcRow_gpu = (int *)malloc(sizeof(int) * num_rows);
        expertForExpandedSrcRow_gpu = (int *)malloc(sizeof(int) * num_rows);
    }

    int cur_expert = 0;
    for (int i = 0; i < expertForSrcRow.size() / k; ++i) {
        for (int j = 0; j < k; ++j) {
            int selected_expert = -1;
            if (randomize_output) {
                do {
                    selected_expert = rand() % num_experts;
                } while (expertCount[selected_expert] > 0);
            } else {
                selected_expert = cur_expert % num_experts;
                cur_expert += 1;
            }
            expertForSrcRow[i + j * num_src_rows] = {selected_expert, i + j * num_src_rows};
            expertCount[selected_expert] += 1;
            totalExpertCount[selected_expert] += 1;
        }
        for (int j = 0; j < k; ++j) {
            expertCount[std::get<0>(expertForSrcRow[i + j * num_src_rows])] = 0;
        }
    }
    expertOffsetsCpu[0] = totalExpertCount[0];
    for (int i = 1; i < num_experts; ++i) {
        expertOffsetsCpu[i] = expertOffsetsCpu[i - 1] + totalExpertCount[i];
    }

    std::vector<std::pair<int, int>> sortedByExpert(expertForSrcRow);

    std::sort(sortedByExpert.begin(), sortedByExpert.end());

    /*
    for (int i = 0; i < num_experts; ++i) {
        std::cout << totalExpertCount[i] << " ";
    }
    std::cout << "\n";
    */

    /*
     std::cout << "Rank " << mype << ": ";
     for (int i = 0; i < sortedByExpert.size(); ++i) {
         std::cout << "("<< std::get<0>(sortedByExpert[i]) << " ," << std::get<1>(sortedByExpert[i])
     << ")";
     }
     */

    for (int i = 0; i < sortedByExpert.size(); ++i) {
        expertForExpandedSrcRow[i] = std::get<0>(sortedByExpert[i]);
        expandedSrcRow[i] = std::get<1>(sortedByExpert[i]);
    }

    expert_counts_gpu_tmp = expert_counts_gpu;
    for (int i = 0; i < num_blocks; i++) {
        hipMemcpy(expert_counts_gpu_tmp, totalExpertCount.data(), num_experts * sizeof(int64_t),
                   hipMemcpyHostToDevice);
        expert_counts_gpu_tmp += num_experts;
    }
    hipMemcpy(expert_offsets_gpu, expertOffsetsCpu.data(), num_experts * sizeof(int64_t),
               hipMemcpyHostToDevice);
    hipMemcpy(expandedSrcRow_gpu, expandedSrcRow.data(), num_rows * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(expertForExpandedSrcRow_gpu, expertForExpandedSrcRow.data(), num_rows * sizeof(int),
               hipMemcpyHostToDevice);

    set_counts<<<num_src_rows, 1024, 0, stream>>>((float *)send_data, mype, hidden_dim);

    cooperative_launch_args[0] = &send_data;
    cooperative_launch_args[1] = &recv_data;
    cooperative_launch_args[2] = &expandedSrcRow_gpu;
    cooperative_launch_args[3] = &expertForExpandedSrcRow_gpu;
    cooperative_launch_args[4] = &expert_counts_gpu;
    cooperative_launch_args[5] = &expert_offsets_gpu;
    cooperative_launch_args[6] = &k;
    cooperative_launch_args[7] = &num_rows;
    cooperative_launch_args[8] = &mype;
    cooperative_launch_args[9] = &npes;
    cooperative_launch_args[10] = &hidden_dim;
    if (comms_profile < MOE_ONE_STEP_LAUNCH_MIN) {
        if (comms_profile == MOE_COLL_LAUNCH_TWO_STEP_ALLPUSH) {
            cooperative_launch_args[11] = &num_copies_per_block;
            cooperative_launch_args[12] = &expert_pos_out_gpu;
        } else {
            cooperative_launch_args[11] = &expert_pos_out_gpu;
        }
    } else {
        if (comms_profile == MOE_COLL_LAUNCH_ONE_STEP_ALLPUSH) {
            cooperative_launch_args[11] = &num_copies_per_block;
            cooperative_launch_args[12] = &expert_counts_gpu;
        } else {
            cooperative_launch_args[11] = &expert_counts_gpu;
        }
    }

    switch (comms_profile) {
        case MOE_COLL_LAUNCH_ONE_STEP_ALLPUSH:
        case MOE_BARE_ONE_STEP_ALLPUSH:
            if (warp_specialized) {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_one_step_allpush<true>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            } else {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_one_step_allpush<false>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            }
            break;
        case MOE_COLL_LAUNCH_ONE_STEP_BY_PEER:
        case MOE_BARE_ONE_STEP_BY_PEER:
            if (warp_specialized) {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_one_step_by_peer<true>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            } else {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_one_step_by_peer<false>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            }
            break;
        case MOE_BARE_TWO_STEP_ALLPUSH:
        case MOE_COLL_LAUNCH_TWO_STEP_ALLPUSH:
            if (warp_specialized) {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_two_step_allpush<true>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            } else {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_two_step_allpush<false>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            }
            break;
        case MOE_BARE_TWO_STEP_BY_PEER:
        case MOE_COLL_LAUNCH_TWO_STEP_BY_PEER:
            if (warp_specialized) {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_two_step_by_peer<true>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            } else {
                nvshmem_rc = nvshmemx_collective_launch_query_gridsize(
                    (const void *)token_shuffle_two_step_by_peer<false>, threads_per_block,
                    cooperative_launch_args, 2048, &gridsize);
            }
            break;
        default:
            std::cout << "invalid comms profile (" << comms_profile
                      << ") detected. Cannot continue.\n";
            return -1;
    }

    if (nvshmem_rc != NVSHMEMX_SUCCESS) {
        std::cout << "Failed to query for the gridsize of a collective launch API.\n";
        return -1;
    }

    if (gridsize < num_blocks) {
        std::cout << "gridsize (" << gridsize
                  << ") from collective launch query is smaller than requested blocks ("
                  << num_blocks << "). Cannot continue\n";
        return -1;
    }

    switch (comms_profile) {
        case MOE_BARE_TWO_STEP_ALLPUSH:
            if (warp_specialized) {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    token_shuffle_two_step_allpush<true>
                        <<<num_blocks, threads_per_block, 0, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            num_copies_per_block, expert_pos_out_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    token_shuffle_two_step_allpush<false>
                        <<<num_blocks, threads_per_block, 0, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            num_copies_per_block, expert_pos_out_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_COLL_LAUNCH_TWO_STEP_ALLPUSH:
            if (warp_specialized) {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    nvshmemx_collective_launch((const void *)token_shuffle_two_step_allpush<true>,
                                               num_blocks, threads_per_block,
                                               cooperative_launch_args, 0, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    nvshmemx_collective_launch((const void *)token_shuffle_two_step_allpush<false>,
                                               num_blocks, threads_per_block,
                                               cooperative_launch_args, 0, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_BARE_TWO_STEP_BY_PEER:
            if (warp_specialized) {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    token_shuffle_two_step_by_peer<true>
                        <<<num_blocks, threads_per_block, 0, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            expert_counts_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    token_shuffle_two_step_by_peer<false>
                        <<<num_blocks, threads_per_block, 0, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            expert_counts_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_COLL_LAUNCH_TWO_STEP_BY_PEER:
            if (warp_specialized) {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    nvshmemx_collective_launch((const void *)token_shuffle_two_step_by_peer<true>,
                                               num_blocks, threads_per_block,
                                               cooperative_launch_args, 0, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    exchange_offsets<<<1, num_experts * npes, offset_exchange_shared_memory,
                                       stream>>>(expert_counts_gpu, expert_pos_out_gpu, npes);
                    nvshmemx_collective_launch((const void *)token_shuffle_two_step_by_peer<false>,
                                               num_blocks, threads_per_block,
                                               cooperative_launch_args, 0, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_BARE_ONE_STEP_ALLPUSH:
            if (warp_specialized) {
                run = [&]() {
                    token_shuffle_one_step_allpush<true>
                        <<<num_blocks, threads_per_block, one_step_shared_memory, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            num_copies_per_block, expert_counts_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    token_shuffle_one_step_allpush<false>
                        <<<num_blocks, threads_per_block, one_step_shared_memory, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            num_copies_per_block, expert_counts_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_COLL_LAUNCH_ONE_STEP_ALLPUSH:
            if (warp_specialized) {
                run = [&]() {
                    nvshmemx_collective_launch(
                        (const void *)token_shuffle_one_step_allpush<true>, num_blocks,
                        threads_per_block, cooperative_launch_args, one_step_shared_memory, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    nvshmemx_collective_launch(
                        (const void *)token_shuffle_one_step_allpush<false>, num_blocks,
                        threads_per_block, cooperative_launch_args, one_step_shared_memory, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_BARE_ONE_STEP_BY_PEER:
            if (warp_specialized) {
                run = [&]() {
                    token_shuffle_one_step_by_peer<true>
                        <<<num_blocks, threads_per_block, one_step_shared_memory, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            expert_counts_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    token_shuffle_one_step_by_peer<false>
                        <<<num_blocks, threads_per_block, one_step_shared_memory, stream>>>(
                            send_data, recv_data, expandedSrcRow_gpu, expertForExpandedSrcRow_gpu,
                            expert_offsets_gpu, k, num_rows, mype, npes, hidden_dim,
                            expert_counts_gpu);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        case MOE_COLL_LAUNCH_ONE_STEP_BY_PEER:
            if (warp_specialized) {
                run = [&]() {
                    nvshmemx_collective_launch(
                        (const void *)token_shuffle_one_step_by_peer<true>, num_blocks,
                        threads_per_block, cooperative_launch_args, one_step_shared_memory, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            } else {
                run = [&]() {
                    nvshmemx_collective_launch(
                        (const void *)token_shuffle_one_step_by_peer<false>, num_blocks,
                        threads_per_block, cooperative_launch_args, one_step_shared_memory, stream);
                    nvshmemx_barrier_all_on_stream(stream);
                };
            }
            break;
        default:
            std::cout << "invalid comms profile (" << comms_profile
                      << ") detected. Cannot continue.\n";
            return -1;
    }

    for (int i = 0; i < 5; ++i) {
        run();
    }
    nvshmemx_barrier_all_on_stream(stream);
    CUDA_CHECK(hipDeviceSynchronize());
    hipEventRecord(start, stream);
    for (int i = 0; i < iterations; ++i) {
        run();
    }
    hipEventRecord(stop, stream);

    CUDA_CHECK(hipStreamSynchronize(stream));
    hipEventElapsedTime(&milliseconds, start, stop);
    CUDA_CHECK(hipDeviceSynchronize());

    std::cout << "Rank: " << mype << " Time: " << (milliseconds * 1000) / iterations << "\n";

    std::vector<float> recv_data_cpu(expert_factor * num_rows * hidden_dim);

    hipMemcpy(recv_data_cpu.data(), recv_data, num_rows * hidden_dim * sizeof(float),
               hipMemcpyDeviceToHost);

    /*
    std::cout << "Rank " << mype << ": ";
    for (int i = 0; i < num_rows; ++i) {
        std::cout << std::setprecision(10) << recv_data_cpu[i * hidden_dim + 512] << " ";
    }
    std::cout << "\n";
    */

    nvshmem_free(send_data);
    nvshmem_free(recv_data);

    nvshmem_finalize();

    return 0;
}
